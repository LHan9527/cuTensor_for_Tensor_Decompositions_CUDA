#include "head.h"
#include <hipblas.h>

void tucker(float* X,float *G,float *h_A1,float *h_A2,float *h_A3,int I,int J,int K){
	int P = 10;
	int Q = 10;
	int R = 10;
	size_t sd = sizof(float);


float *d_X; 
hipMalloc((void **)&d_X, I*J*K*sd);
hipblasSetVector(I*J*K,sd,X,1,d_X,1);

float *d_G; 
hipMalloc((void **)&d_G,P*Q*R*sd);

float *d_A1, *d_A2, *d_A3;
   hipMalloc((void **)&d_A1, I*P*sd);
   hipMalloc((void **)&d_A2, J*Q*sd);
   hipMalloc((void **)&d_A3, K*R*sd);

   hipblasSetVector(I*P,sd,h_A1,1,d_A1,1);
   hipblasSetVector(J*Q,sd,h_A2,1,d_A2,1);
   hipblasSetVector(K*R,sd,h_A3,1,d_A3,1);

float *d_temp_3;
 hipMalloc((void **)&d_temp_3,P*J*K*sd);

 float *d_temp_4;
 hipMalloc((void **)&d_temp_4,P*J*K*sd);

hipblasHandle_t handle; hipblasCreate(&handle);  
  float alpha = 1.0f, beta = 0.0f;

  hipblasSgemmBatched(handle, HIPBLAS_OP_T,HIPBLAS_OP_N,P,J,I,&alpha,d_A1,I,0,d_X,I,I*J,&beta,d_temp_3,P,P*J,K);
  hipblasSgemmBatched(handle, HIPBLAS_OP_N,HIPBLAS_OP_N,P,J,K,&alpha,d_temp_3,P,P*J,d_A3,K,0,&beta,d_temp_4,P,P*J,R);
  hipblasSgemmBatched(handle, HIPBLAS_OP_N,HIPBLAS_OP_N,P,Q,J,&alpha,d_temp_4,P,P*J,d_A2,J,0,&beta,d_G,P,P*Q,R);

  hipblasGetVector(P*Q*R,sd,d_G,1,G,1); 

hipFree(d_A1);hipFree(d_A2);hipFree(d_A3);
hipFree(d_temp_3);hipFree(d_temp_4)
  hipFree(d_G);
hipFree(d_X);
  hipblasDestroy(handle);
}
