#include "opera.h"
void cuinv(dt *A,dt *B,int m){
	dt* d_A;
	dt* d_B;
	hipMalloc((void**)&d_B,sizeof(dt)*m*m);
	hipMalloc((void**)&d_A,sizeof(dt)*m*m);
	hipMemcpy(d_A,A,sizeof(dt)*m*m,hipMemcpyHostToDevice);
	dt *d_U;
	dt *d_S;
	dt *d_V;
	dt *U = new dt[m*m]();
	dt *S = new dt[m]();
	dt *V = new dt[m*m]();
	int *d_info = NULL;
	int lwork = 0;
	dt *d_work = NULL;
	int info = 0;
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	const int econ = 0;

	hipsolverHandle_t cusolverH = NULL;
	hipsolverGesvdjInfo_t gesvdj_params = NULL;

	hipsolverDnCreate(&cusolverH);
	hipsolverDnCreateGesvdjInfo(&gesvdj_params);

	hipMalloc((void**)&d_U,sizeof(dt)*m*m);	
	hipMalloc((void**)&d_S,sizeof(dt)*m);	
	hipMalloc((void**)&d_V,sizeof(dt)*m*m);	
	hipMalloc((void**)&d_info,sizeof(int));
	hipsolverDnSgesvdj_bufferSize(
			cusolverH,
			jobz,
			econ,
			m,
			m,
			d_A,
			m,
			d_S,
			d_U,
			m,
			d_V,
			m,
			&lwork,
			gesvdj_params
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgesvdj(
			cusolverH,
			jobz,
			econ,
			m,
			m,
			d_A,
			m,
			d_S,
			d_U,
			m,
			d_V,
			m,
			d_work,
			lwork,
			d_info,
			gesvdj_params

			);
	hipDeviceSynchronize();
	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);
	if(0 == info){
		cout<<"ok"<<endl;
	}else if(0>info){
		cout<<-info<<"is wrong"<<endl;
	}else{
		cout<<info<<"do not work"<<endl;
	}
	hipFree(d_A);

//	hipMemcpy(U,d_U,sizeof(dt)*m*m,hipMemcpyDeviceToHost);
//	hipMemcpy(S,d_S,sizeof(dt)*m,hipMemcpyDeviceToHost);
//	hipMemcpy(V,d_V,sizeof(dt)*m*m,hipMemcpyDeviceToHost);

//	printTensor(U,m,m,1);
//	printTensor(S,m,1,1);
//	printTensor(V,m,m,1);
	
	hipDeviceSynchronize();
	
	dim3 threads(512,1,1);
	dim3 blocks((m*m+512-1)/512,1,1);
	matvec<<<blocks,threads>>>(d_U,d_S,m);
	hipFree(d_S);

	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasStatus_t cublasStat = hipblasCreate(&handle);
	hipblasSgemm(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			m,
			m,
			m,
			&alpha,
			d_U,
			m,
			d_V,
			m,
			&beta,
			d_B,  //store A*A'
			m
			);
	hipFree(d_U);
	hipFree(d_V);
	dt *d_BT;
	hipMalloc((void**)&d_BT,sizeof(dt)*m*m);
	transpose<<<blocks,threads>>>(d_B,d_BT,m,m);

	hipMemcpy(B,d_BT,sizeof(dt)*m*m,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_B);
	hipFree(d_BT);

	hipsolverDnDestroy(cusolverH);
	hipsolverDnDestroyGesvdjInfo(gesvdj_params);
	hipDeviceReset();
	
	delete[] U;U = nullptr;
	delete[] S;S = nullptr;
	delete[] V;V = nullptr;

}
