#include "opera.h"

void maxpro(dt *A,dt *B,dt *C,int a,int b,int c){
	// A a*b, B b*c  C a*c
	dt *d_A;
	dt *d_B;
	dt *d_C;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b);
	hipMalloc((void**)&d_B,sizeof(dt)*b*c);
	hipMalloc((void**)&d_C,sizeof(dt)*a*c);

	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c,hipMemcpyHostToDevice);
	
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasStatus_t cublasStat = hipblasCreate(&handle);
	hipblasSgemm(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			c,
			a,
			b,
			&alpha,
			d_B,
			c,
			d_A,
			b,
			&beta,
			d_C,  //store A*A'
			c
			);

	hipMemcpy(C,d_C,sizeof(dt)*a*c,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

void V100maxpro(dt *A,dt *B,dt *C,int a,int b,int c){
	hipblasHandle_t handle;
	hipblasStatus_t cublasStat = hipblasCreate(&handle);
	cublasStat = hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	
	dt *d_A;
	dt *d_B;
	dt *d_C;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b);
	hipMalloc((void**)&d_B,sizeof(dt)*b*c);
	hipMalloc((void**)&d_C,sizeof(dt)*a*c);

	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c,hipMemcpyHostToDevice);
//	hipblasGemmAlgo_t algo;
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			c,
			a,
			b,
			&alpha,
			d_B,
			HIP_R_16F,
			c,
			d_A,
			HIP_R_16F,
			b,
			&beta,
			d_C,
			HIP_R_16F,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);

	hipMemcpy(C,d_C,sizeof(dt)*a*c,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
 
void v100mpStride(dt *A,dt *B,dt *C,int a,int b,int c,int r){
	//  A a*b*c   B b*r  C a*r*b
	hipblasHandle_t handle;
	hipblasStatus_t cublasStat = hipblasCreate(&handle);
	cublasStat = hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	
	dt *d_A;
	dt *d_B;
	dt *d_C;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c);
	hipMalloc((void**)&d_B,sizeof(dt)*b*r);
	hipMalloc((void**)&d_C,sizeof(dt)*a*r*c);

	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*r,hipMemcpyHostToDevice);
	
	dt *temp = new dt[a*b*c]();
	dt *temp1 = new dt[a*b*c]();
	dt *d_temp;
	dt *d_temp1;
	hipMalloc((void**)&d_temp,sizeof(dt)*a*b*c);
	hipMalloc((void**)&d_temp1,sizeof(dt)*a*b*c);
	dim3 threads(512,1,1);
	dim3 blocks((a*b*c+512-1)/512,1,1);

	mode3tran<<<blocks,threads>>>(d_A,d_temp,a,b,c);
//	hipMemcpy(temp,d_temp,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);

	tran3mode<<<blocks,threads>>>(d_temp,d_temp1,a,b,c);
	hipMemcpy(temp1,d_temp1,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);

//	printTensor(temp,c,a*b,1);
	printTensor(temp1,a,b,c);
	hipFree(d_temp);
	hipFree(d_temp1);
	delete[] temp;temp = nullptr;
	

	dt beta = 1.0;
	dt alpha = 0.0;

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r,
			a,
			b,
			&alpha,
			d_B,
			HIP_R_16F,
			r,
			0,
			d_A,
			HIP_R_16F,
			b,
			b*a,
			&beta,
			d_C,
			HIP_R_16F,
			r,
			r*a,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT

			);

	hipMemcpy(C,d_C,sizeof(dt)*a*r*c,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}
