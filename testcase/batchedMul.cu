#include "hip/hip_runtime.h"
#include "head.h"


void setMatrice(dt *A,int m,int n){
	for(int i = 0;i<m*n;i++){
		A[i] = rand()*0.1/(RAND_MAX*0.1);
	}
}
int main(int args,char *argv[]){
	int a = atoi(argv[1]);
	int b = atoi(argv[2]);
	int c = atoi(argv[3]);

	clock_t t1,t2,t3,t4;
	t1=clock();
	dt **h_A = new dt*[c];
	dt **h_B = new dt*[c]; //host CPU
	dt **h_C = new dt*[c];
	dt **h_result = new dt*[c];
	for(int i = 0;i<c;i++){
		h_result[i] = new dt[a*a];
		hipMalloc((void**)&h_A[i],sizeof(dt)*a*b);
		hipMalloc((void**)&h_B[i],sizeof(dt)*b*a);
		hipMalloc((void**)&h_C[i],sizeof(dt)*a*a);
		// malloc memory on GPU and link to host
	}//link device to host

	dt **d_A;
	dt **d_B;
	dt **d_C;
	hipMalloc((void**)&d_A,sizeof(*h_A)*c);
	hipMalloc((void**)&d_B,sizeof(*h_B)*c);
	hipMalloc((void**)&d_C,sizeof(*h_C)*c);

	hipMemcpy(d_A,h_A,sizeof(*h_A)*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,sizeof(*h_B)*c,hipMemcpyHostToDevice);
	hipMemcpy(d_C,h_C,sizeof(*h_C)*c,hipMemcpyHostToDevice);
	//Device malloc memory and transfer Host to Device

	dt *A = new dt[a*b]();
	dt *B = new dt[b*a]();
	dt *C = new dt[a*a]();
	for(int i =0;i<c;i++){
		setMatrice(A,a,b);
//		printTensor(A,a,b,1);
		hipblasSetMatrix(a,b,sizeof(dt),A,a,h_A[i],a);
		setMatrice(B,b,a);
//		printTensor(B,b,a,1);
		hipblasSetMatrix(b,a,sizeof(dt),B,b,h_B[i],b);
	}

	const dt **AA =(const dt **)d_A;
	const dt **BB =(const dt **)d_B;
	t3 = clock();
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemmBatched(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		a,
		a,
		b,
		&alpha,
		BB,
		a,
		AA,
		b,
		&beta,
		d_C,
		a,
		c
		);
	t4 = clock();
	for(int i = 0;i<c;i++){
		hipblasGetMatrix(a,a,sizeof(dt),h_C[i],a,h_result[i],a);
//		printTensor(h_result[i],a,a,1);
	}
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	delete[] h_A;h_A = NULL;
	delete[] h_B;h_B = NULL;
	delete[] h_C;h_C = NULL;
	delete[] h_result;h_result = NULL;
	t2 = clock();
	cout<<(double)(t4-t3)/CLOCKS_PER_SEC<<"s"<<endl;
	cout<<(double)(t2-t1)/CLOCKS_PER_SEC<<"s"<<endl;
	return 0;
}

