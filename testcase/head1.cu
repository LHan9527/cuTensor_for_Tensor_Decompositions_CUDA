#include "hip/hip_runtime.h"
#include "head.h"

void cuStrideModetran(dt *A,dt *B,dt *res,int a,int b,int c){
	dt *d_A = NULL;	
	dt *d_BT = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
//	dt *BT = new dt[b*a]();
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c); 	//a*b	
	hipMalloc((void**)&d_BT,sizeof(dt)*b*a); 	//a*b	
	hipMalloc((void**)&d_B,sizeof(dt)*a*b);	//a*c	
	hipMalloc((void**)&d_res,sizeof(dt)*a*a*c);	//b*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*a,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgeam(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		a,
		b,
		&alpha,
		d_B,
		b,
		&beta,
		d_BT,
		a,
		d_BT,
		a
		 ); //d_BT is b*a d_A is a*b*c d_res is a*a*c
//	hipMemcpy(BT,d_BT,sizeof(dt)*b*a,hipMemcpyDeviceToHost);
//	printTensor(BT,b,a,1);
	hipblasSgemmStridedBatched(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		a,
		a,
		b,
		&alpha,
		d_BT,
		a,
		0,
		d_A,
		b,
		b*a,
		&beta,
		d_res,
		a,
		a*a,
		c
	           );

	hipMemcpy(res,d_res,sizeof(dt)*a*a*c,hipMemcpyDeviceToHost);
//	printTensor(res,a,a,c);
	hipblasDestroy(handle);
//	delete[] BT;BT = nullptr;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);
	hipFree(d_BT);

}

void cuStrideModenotran(dt *A,dt *B,dt *res,int a,int b,int c){
	// A is a*b*c B is b*c res is a*c*c
	dt *d_A = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c); 	//a*b*c	
	hipMalloc((void**)&d_B,sizeof(dt)*b*a);	//b*c	
	hipMalloc((void**)&d_res,sizeof(dt)*a*a*c);	//a*c*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*a,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
//	clock_t time1,time2;
//	time1 = clock();
// d_A a*b*c d_B a*b res is a*a*c
	hipblasSgemmStridedBatched(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		a,
		a,
		b,
		&alpha,
		d_B,
		b,
		0,
		d_A,
		b,
		b*a,
		&beta,
		d_res,
		a,
		a*a,
		c
		);

//	time2 = clock();
//	cout<<(double)(time2-time1)/CLOCKS_PER_SEC<<"s"<<endl;
	hipMemcpy(res,d_res,sizeof(dt)*a*a*c,hipMemcpyDeviceToHost);
//	printTensor(res,a,a,c);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);
}
