#include "hip/hip_runtime.h"
#include "head.h"

void newtest(dt *A,dt *U1,dt *U2,dt *U3,dt *res1,int a,int b,int c){
	int m = a*0.1;
	int n = b*0.1;
	int k = c*0.1;
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c);
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	dt *d_U1;
	hipMalloc((void**)&d_U1,sizeof(dt)*a*m);
	hipMemcpy(d_U1,U1,sizeof(dt)*a*m,hipMemcpyHostToDevice);
	dt *d_temp1;
	hipMalloc((void**)&d_temp1,sizeof(dt)*m*b*c);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			b,
			m,
			a,
			&alpha,
			d_A,
			b,
			b*a,
			d_U1,
			m,
			0,
			&beta,
			d_temp1,
			b,
			b*m,
			c
			);
	// now d_temp1 is b*m*c row storage
	hipFree(d_U1);
	hipFree(d_A);
	dt *d_U2;
	hipMalloc((void**)&d_U2,sizeof(dt)*b*n);
	hipMemcpy(d_U2,U2,sizeof(dt)*b*n,hipMemcpyHostToDevice);
	dt *d_temp2;
	hipMalloc((void**)&d_temp2,sizeof(dt)*m*n*c);

	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			n,m,b,
			&alpha,
			d_U2,
			n,
			0,
			d_temp1,
			b,
			b*m,
			&beta,
			d_temp2,
			n,
			n*m,
			c
			);
	// now d_temp3 is m*n*c row storage
	hipFree(d_U2);
	hipFree(d_temp1);
	dt *d_U3;
	hipMalloc((void**)&d_U3,sizeof(dt)*c*k);
	hipMemcpy(d_U3,U3,sizeof(dt)*c*k,hipMemcpyHostToDevice);
	dt *d_res1;
	hipMalloc((void**)&d_res1,sizeof(dt)*m*n*k);

	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			1,n*n,c,
			&alpha,
			d_U3,
			1,
			c,
			d_temp2,
			c,
			0,
			&beta,
			d_res1,
			1,
			m*n,
			k
			);
	// now d_res1 is m*n*k row storage
	hipMemcpy(res1,d_res1,sizeof(dt)*m*n*k,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_U3);
	hipFree(d_res1);

	
}
void newtest32(dt *A,dt *U1,dt *U2,dt *U3,dt *res1,int a,int b,int c){
	int m = a*0.1;
	int n = b*0.1;
	int k = c*0.1;
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c);
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	dt *d_U1;
	hipMalloc((void**)&d_U1,sizeof(dt)*a*m);
	hipMemcpy(d_U1,U1,sizeof(dt)*a*m,hipMemcpyHostToDevice);
	dt *d_temp1;
	hipMalloc((void**)&d_temp1,sizeof(dt)*m*b*c);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			b,
			m,
			a,
			&alpha,
			d_A,HIP_R_32F,
			b,
			b*a,
			d_U1,HIP_R_32F,
			m,
			0,
			&beta,
			d_temp1,HIP_R_32F,
			b,
			b*m,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_temp1 is b*m*c row storage
	hipFree(d_U1);
	hipFree(d_A);
	dt *d_U2;
	hipMalloc((void**)&d_U2,sizeof(dt)*b*n);
	hipMemcpy(d_U2,U2,sizeof(dt)*b*n,hipMemcpyHostToDevice);
	dt *d_temp2;
	hipMalloc((void**)&d_temp2,sizeof(dt)*m*n*c);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			n,m,b,
			&alpha,
			d_U2,HIP_R_32F,
			n,
			0,
			d_temp1,HIP_R_32F,
			b,
			b*m,
			&beta,
			d_temp2,HIP_R_32F,
			n,
			n*m,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_temp3 is m*n*c row storage
	hipFree(d_U2);
	hipFree(d_temp1);
	dt *d_U3;
	hipMalloc((void**)&d_U3,sizeof(dt)*c*k);
	hipMemcpy(d_U3,U3,sizeof(dt)*c*k,hipMemcpyHostToDevice);
	dt *d_res1;
	hipMalloc((void**)&d_res1,sizeof(dt)*m*n*k);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			1,n*n,c,
			&alpha,
			d_U3,HIP_R_32F,
			1,
			c,
			d_temp2,HIP_R_32F,
			c,
			0,
			&beta,
			d_res1,HIP_R_32F,
			1,
			m*n,
			k,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_res1 is m*n*k row storage
	hipMemcpy(res1,d_res1,sizeof(dt)*m*n*k,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_U3);
	hipFree(d_res1);

	
}

void newtest16(dt *A,dt *U1,dt *U2,dt *U3,dt *res1,int a,int b,int c){
	int m = a*0.1;
	int n = b*0.1;
	int k = c*0.1;
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c);
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	dt *d_U1;
	hipMalloc((void**)&d_U1,sizeof(dt)*a*m);
	hipMemcpy(d_U1,U1,sizeof(dt)*a*m,hipMemcpyHostToDevice);
	dt *d_temp1;
	hipMalloc((void**)&d_temp1,sizeof(dt)*m*b*c);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	half *h_A;
	hipMalloc((void**)&h_A,sizeof(half)*a*b*c);
	half *h_U1;
	hipMalloc((void**)&h_U1,sizeof(half)*a*m);
	dim3 threads(512,1,1);
	dim3 block1((a*b*c/512+512-1),1,1);
	f2h<<<block1,threads>>>(d_A,h_A,a*b*c);
	dim3 block2((a*m/512+512-1),1,1);
	f2h<<<block2,threads>>>(d_U1,h_U1,a*m);
	hipFree(d_U1);
	hipFree(d_A);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			b,
			m,
			a,
			&alpha,
			h_A,
			HIP_R_16F,
			b,
			b*a,
			h_U1,
			HIP_R_16F,
			m,
			0,
			&beta,
			d_temp1,
			HIP_R_32F,
			b,
			b*m,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_temp1 is b*m*c row storage
	hipFree(h_U1);
	hipFree(h_A);
	dt *d_U2;
	hipMalloc((void**)&d_U2,sizeof(dt)*b*n);
	hipMemcpy(d_U2,U2,sizeof(dt)*b*n,hipMemcpyHostToDevice);
	half *h_U2;
	hipMalloc((void**)&h_U2,sizeof(half)*b*n);
	half *h_temp1;
	hipMalloc((void**)&h_temp1,sizeof(half)*b*m*c);
	dim3 block3((m*b*c/512+512-1),1,1);
	f2h<<<block3,threads>>>(d_temp1,h_temp1,m*b*c);
	dim3 block4((b*n/512+512-1),1,1);
	f2h<<<block4,threads>>>(d_U2,h_U2,b*n);
	hipFree(d_U2);
	hipFree(d_temp1);
	dt *d_temp2;
	hipMalloc((void**)&d_temp2,sizeof(dt)*m*n*c);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			n,m,b,
			&alpha,
			h_U2,
			HIP_R_16F,
			n,
			0,
			h_temp1,
			HIP_R_16F,
			b,
			b*m,
			&beta,
			d_temp2,
			HIP_R_32F,
			n,
			n*m,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_temp3 is m*n*c row storage
	hipFree(h_U2);
	hipFree(h_temp1);
	dt *d_U3;
	hipMalloc((void**)&d_U3,sizeof(dt)*c*k);
	hipMemcpy(d_U3,U3,sizeof(dt)*c*k,hipMemcpyHostToDevice);

	half *h_U3;
	hipMalloc((void**)&h_U3,sizeof(half)*c*k);
	half *h_temp2;
	hipMalloc((void**)&h_temp2,sizeof(half)*n*m*c);
	dim3 block5((c*k/512+512-1),1,1);
	f2h<<<block5,threads>>>(d_U3,h_U3,c*k);
	dim3 block6((m*n*c/512+512-1),1,1);
	f2h<<<block6,threads>>>(d_temp2,h_temp2,m*n*c);
	hipFree(d_U3);
	hipFree(d_temp2);
	dt *d_res1;
	hipMalloc((void**)&d_res1,sizeof(dt)*m*n*k);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			1,n*n,c,
			&alpha,
			h_U3,
			HIP_R_16F,
			1,
			c,
			h_temp2,
			HIP_R_16F,
			c,
			0,
			&beta,
			d_res1,
			HIP_R_32F,
			1,
			m*n,
			k,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_res1 is m*n*k row storage
	hipMemcpy(res1,d_res1,sizeof(dt)*m*n*k,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(h_U3);
	hipFree(h_temp2);
	hipFree(d_res1);

	
}


void newtest16h(dt *A,dt *U1,dt *U2,dt *U3,dt *res1,int a,int b,int c){
	int m = a*0.1;
	int n = b*0.1;
	int k = c*0.1;
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c);
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	dt *d_U1;
	hipMalloc((void**)&d_U1,sizeof(dt)*a*m);
	hipMemcpy(d_U1,U1,sizeof(dt)*a*m,hipMemcpyHostToDevice);
	half *d_temp1;
	hipMalloc((void**)&d_temp1,sizeof(half)*m*b*c);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	half *h_A;
	hipMalloc((void**)&h_A,sizeof(half)*a*b*c);
	half *h_U1;
	hipMalloc((void**)&h_U1,sizeof(half)*a*m);
	dim3 threads(512,1,1);
	dim3 block1((a*b*c/512+512-1),1,1);
	f2h<<<block1,threads>>>(d_A,h_A,a*b*c);
	dim3 block2((a*m/512+512-1),1,1);
	f2h<<<block2,threads>>>(d_U1,h_U1,a*m);
	hipFree(d_U1);
	hipFree(d_A);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			b,
			m,
			a,
			&alpha,
			h_A,
			HIP_R_16F,
			b,
			b*a,
			h_U1,
			HIP_R_16F,
			m,
			0,
			&beta,
			d_temp1,
			HIP_R_16F,
			b,
			b*m,
			c,
			HIP_R_16F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_temp1 is b*m*c row storage
	hipFree(h_U1);
	hipFree(h_A);
	dt *d_U2;
	hipMalloc((void**)&d_U2,sizeof(dt)*b*n);
	hipMemcpy(d_U2,U2,sizeof(dt)*b*n,hipMemcpyHostToDevice);
	half *h_U2;
	hipMalloc((void**)&h_U2,sizeof(half)*b*n);
	dim3 block4((b*n/512+512-1),1,1);
	f2h<<<block4,threads>>>(d_U2,h_U2,b*n);
	hipFree(d_U2);
	half *d_temp2;
	hipMalloc((void**)&d_temp2,sizeof(half)*m*n*c);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			n,m,b,
			&alpha,
			h_U2,
			HIP_R_16F,
			n,
			0,
			d_temp1,
			HIP_R_16F,
			b,
			b*m,
			&beta,
			d_temp2,
			HIP_R_16F,
			n,
			n*m,
			c,
			HIP_R_16F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_temp3 is m*n*c row storage
	hipFree(h_U2);
	hipFree(d_temp1);
	dt *d_U3;
	hipMalloc((void**)&d_U3,sizeof(dt)*c*k);
	hipMemcpy(d_U3,U3,sizeof(dt)*c*k,hipMemcpyHostToDevice);

	half *h_U3;
	hipMalloc((void**)&h_U3,sizeof(half)*c*k);
	dim3 block5((c*k/512+512-1),1,1);
	f2h<<<block5,threads>>>(d_U3,h_U3,c*k);
	hipFree(d_U3);
	dt *d_res1;
	hipMalloc((void**)&d_res1,sizeof(dt)*m*n*k);

	hipblasGemmStridedBatchedEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			1,n*n,c,
			&alpha,
			h_U3,
			HIP_R_16F,
			1,
			c,
			d_temp2,
			HIP_R_16F,
			c,
			0,
			&beta,
			d_res1,
			HIP_R_32F,
			1,
			m*n,
			k,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	// now d_res1 is m*n*k row storage
	hipMemcpy(res1,d_res1,sizeof(dt)*m*n*k,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(h_U3);
	hipFree(d_temp2);
	hipFree(d_res1);

	
}





void cuStrideModetran(dt *A,dt *B,dt *res,int a,int b,int c){
	dt *d_A = NULL;	
	dt *d_BT = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
//	dt *BT = new dt[b*a]();
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c); 	//a*b	
	hipMalloc((void**)&d_BT,sizeof(dt)*b*a); 	//a*b	
	hipMalloc((void**)&d_B,sizeof(dt)*a*b);	//a*c	
	hipMalloc((void**)&d_res,sizeof(dt)*a*a*c);	//b*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*a,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgeam(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		a,
		b,
		&alpha,
		d_B,
		b,
		&beta,
		d_BT,
		a,
		d_BT,
		a
		 ); //d_BT is b*a d_A is a*b*c d_res is a*a*c
//	hipMemcpy(BT,d_BT,sizeof(dt)*b*a,hipMemcpyDeviceToHost);
//	printTensor(BT,b,a,1);
	hipblasSgemmStridedBatched(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		a,
		a,
		b,
		&alpha,
		d_BT,
		a,
		0,
		d_A,
		b,
		b*a,
		&beta,
		d_res,
		a,
		a*a,
		c
	           );

	hipMemcpy(res,d_res,sizeof(dt)*a*a*c,hipMemcpyDeviceToHost);
//	printTensor(res,a,a,c);
	hipblasDestroy(handle);
//	delete[] BT;BT = nullptr;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);
	hipFree(d_BT);

}

void cuStrideModenotran(dt *A,dt *B,dt *res,int a,int b,int c){
	// A is a*b*c B is b*c res is a*c*c
	dt *d_A = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c); 	//a*b*c	
	hipMalloc((void**)&d_B,sizeof(dt)*b*a);	//b*c	
	hipMalloc((void**)&d_res,sizeof(dt)*a*a*c);	//a*c*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*a,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
//	clock_t time1,time2;
//	time1 = clock();
// d_A a*b*c d_B a*b res is a*a*c
	hipblasSgemmStridedBatched(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		a,
		a,
		b,
		&alpha,
		d_B,
		b,
		0,
		d_A,
		b,
		b*a,
		&beta,
		d_res,
		a,
		a*a,
		c
		);

//	time2 = clock();
//	cout<<(double)(time2-time1)/CLOCKS_PER_SEC<<"s"<<endl;
	hipMemcpy(res,d_res,sizeof(dt)*a*a*c,hipMemcpyDeviceToHost);
//	printTensor(res,a,a,c);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);
}
