#include "hip/hip_runtime.h"
#include "head.h"

void printTensor(dt *A,int a,int b,int c){
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<A[i*a*b+j*b+k]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
}

void tranpro(dt *A,dt *B,dt *res,int a,int b,int c){

	dt sum = 0.0;
	dt *At = new dt[a*b]();
	for(int i = 0;i<a;i++){
		for(int j = 0;j<b;j++){
			At[j*a+i] = A[i*b+j];
		}
	} //At is b*a B is a*c
//	printTensor(At,b,a,1);
	for(int i = 0;i<b;i++){
		for(int j = 0;j<c;j++){
			sum = 0.0;
			for(int k = 0;k<a;k++){
				sum+=At[i*a+k]*B[k*c+j];
			}	
			res[i*c+j] = sum;
		}
	}

//	printTensor(res,b,c,1);
	
	delete[] At;At=nullptr;
}

void notranpro(dt *A,dt *B,dt *res,int a,int b,int c){
//A a*b B a*c res b*c
	dt sum = 0.0;

	for(int i = 0;i<b;i++){
		for(int j = 0;j<c;j++){
			sum = 0.0;
			for(int k = 0;k<a;k++){
				sum+=A[k*b+i]*B[k*c+j];
			}	
			res[i*c+j] = sum;
		}
	}

//	printTensor(res,b,c,1);
	
}

void cunoTran(dt *A,dt *B,dt *res,int a,int b,int c){
	dt *d_A = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b); 	//a*b	
	hipMalloc((void**)&d_B,sizeof(dt)*a*c);	//a*c	
	hipMalloc((void**)&d_res,sizeof(dt)*c*b);	//b*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*a*c,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_T,
		c,
		b,
		a,
		&alpha,
		d_B,
		c,
		d_A,
		b,
		&beta,
		d_res,
		c
	           );

	hipMemcpy(res,d_res,sizeof(dt)*c*b,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);
}
void cuTran(dt *A,dt *B,dt *res,int a,int b,int c){
	dt *d_A = NULL;	
	dt *d_AT = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b); 	//a*b	
	hipMalloc((void**)&d_AT,sizeof(dt)*a*b); 	//a*b	
	hipMalloc((void**)&d_B,sizeof(dt)*a*c);	//a*c	
	hipMalloc((void**)&d_res,sizeof(dt)*c*b);	//b*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*a*c,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgeam(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		a,
		b,
		&alpha,
		d_A,
		b,
		&beta,
		d_AT,
		a,
		d_AT,
		a
		 );

	hipblasSgemm(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		c,
		b,
		a,
		&alpha,
		d_B,
		c,
		d_AT,
		a,
		&beta,
		d_res,
		c
	           );

	hipMemcpy(res,d_res,sizeof(dt)*c*b,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);
	hipFree(d_AT);

}

void cuStrideMode(dt *A,dt *B,dt *res,int a,int b,int c){
	// A is a*b*c B is b*c res is a*c*c
	dt *d_A = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c); 	//a*b*c	
	hipMalloc((void**)&d_B,sizeof(dt)*b*c);	//b*c	
	hipMalloc((void**)&d_res,sizeof(dt)*a*c*c);	//a*c*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	clock_t time1,time2;
	time1 = clock();

	hipblasSgemmStridedBatched(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		a,
		c,
		b,
		&alpha,
		d_A,
		a,
		a*b,
		d_B,
		b,
		0,
		&beta,
		d_res,
		a,
		a*c,
		c
		);

	time2 = clock();
	cout<<(double)(time2-time1)/CLOCKS_PER_SEC<<"s"<<endl;
	hipMemcpy(res,d_res,sizeof(dt)*a*c*c,hipMemcpyDeviceToHost);
	printTensor(res,a,c,c);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);
	
}

void cuStride(dt *A,dt *B,dt *res,int a,int b,int c){
	// A is a*b*c B is b*c res is a*c*c
	dt *d_A = NULL;	
	dt *d_B = NULL;	
	dt *d_res = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b*c); 	//a*b*c	
	hipMalloc((void**)&d_B,sizeof(dt)*b*c*c);	//b*c	
	hipMalloc((void**)&d_res,sizeof(dt)*a*c*c);	//a*c*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c*c,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	clock_t time1,time2;
	time1 = clock();

	hipblasSgemmStridedBatched(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		a,
		c,
		b,
		&alpha,
		d_A,
		a,
		a*b,
		d_B,
		b,
		b*c,
		&beta,
		d_res,
		a,
		a*c,
	        c
		);

	time2 = clock();
	cout<<(double)(time2-time1)/CLOCKS_PER_SEC<<"s"<<endl;
	hipMemcpy(res,d_res,sizeof(dt)*a*c*c,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_res);

}
