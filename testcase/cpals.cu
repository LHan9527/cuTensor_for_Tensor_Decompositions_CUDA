#include "hip/hip_runtime.h"
#include "head.h"

void allin(dt *X,dt *M,dt *N,dt *res,int m,int n,int r,int k,int flag){
	// X m*n*k left  m*r right  n*r 
	dt *d_X;
	hipMalloc((void**)&d_X,sizeof(dt)*m*n*k);
	dt *d_X_M;
	hipMalloc((void**)&d_X_M,sizeof(dt)*m*n*k);
	hipMemcpy(d_X,X,sizeof(dt)*m*n*k,hipMemcpyHostToDevice);

	dim3 thread(512,1,1);
	dim3 block((m*n*k+512-1)/512,1,1);
	if(flag == 1){
		mode1tran<<<block,thread>>>(d_X,d_X_M,k,n,m);
	}else if(flag == 2){
		mode2tran<<<block,thread>>>(d_X,d_X_M,n,k,m);
	}else{
		mode3tran<<<block,thread>>>(d_X,d_X_M,n,m,k);
	}
	
	hipFree(d_X);  //d_X_M(flag)
	// now we compute M'*M and N'*N

	dt *d_M;
	hipMalloc((void**)&d_M,sizeof(dt)*m*r); //store with row
	hipMemcpy(d_M,M,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	dt *d_MM;
	hipMalloc((void**)&d_MM,sizeof(dt)*r*r); 

	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipblasSsyrk(
		handle,
		uplo,
		HIPBLAS_OP_N,
		r,m,
		&alpha,
		d_M,r,
		&beta,
		d_MM,r	
		);
//	hipFree(d_M);
	//d_MM is store in half 

	dt *d_N;
	hipMalloc((void**)&d_N,sizeof(dt)*n*r);
	hipMemcpy(d_N,N,sizeof(dt)*n*r,hipMemcpyHostToDevice);
	dt *d_NN;
	hipMalloc((void**)&d_NN,sizeof(dt)*r*r);

	hipblasSsyrk(
		handle,
		uplo,
		HIPBLAS_OP_N,
		r,n,
		&alpha,
		d_N,r,
		&beta,
		d_NN,r	
		);
//	hipFree(d_N);

	dim3 block1((r*r+512-1),1,1);
	elepro<<<block1,thread>>>(d_MM,d_NN,r);
	//d_MM store the half element
/*	dt *tt = new dt[r*r]();
	hipMemcpy(tt,d_MM,sizeof(dt)*r*r,hipMemcpyDeviceToHost);
	printTensor(tt,r,r,1);
	delete[] tt;tt=nullptr;
	cout<<"this is yuansucheng"<<endl;
*/
	dt *d_dot;
	hipMalloc((void **)&d_dot,sizeof(dt)*m*n*r);
	dim3 block2((m*n*r+512-1)/512,1,1);
	krpro<<<block2,thread>>>(d_M,d_N,d_dot,m,n,r);
	hipFree(d_M);
	hipFree(d_N);
/*	dt *t1 = new dt[m*n*r]();
	hipMemcpy(t1,d_dot,sizeof(dt)*m*n*r,hipMemcpyDeviceToHost);
	printTensor(t1,m*n,r,1);
	delete[] t1;t1=nullptr;
	cout<<"this is KR product"<<endl;
*/
	// d_X_M is k*mn d_dot mn*r store with row
	// d_X_M * d_dot

	dt *d_req;
	hipMalloc((void**)&d_req,sizeof(dt)*r*k);
	hipblasSgemm(

		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		r,k,m*n,
		&alpha,
		d_dot,r,
		d_X_M,m*n,
		&beta,
		d_req,r
			);
	hipFree(d_X_M);
	hipFree(d_dot);
	hipblasDestroy(handle);

/*	dt *t2 = new dt[k*r]();
	hipMemcpy(t2,d_req,sizeof(dt)*k*r,hipMemcpyDeviceToHost);
	printTensor(t2,k,r,1);
	delete[] t2;t2=nullptr;
	cout<<"this is left"<<endl;
*/
	//d_req  is the left k*r
	// d_MM is the right r*r 
	// the result is k*r

	dt *d_work;
	int *d_info;
	int lwork;
	hipsolverHandle_t handle1;
	hipsolverDnCreate(&handle1);
	hipMalloc((void**)&d_info,sizeof(int));

	hipsolverDnSpotrf_bufferSize(
			handle1,
			uplo,
			r,
			d_MM,
			r,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);

	hipsolverDnSpotrf(
			handle1,
			uplo,
			r,
			d_MM,
			r,
			d_work,
			lwork,
			d_info
			);
	hipsolverDnSpotrs(
			handle1,
			uplo,
			r,
			k,
			d_MM,
			r,
			d_req,
			r,
			d_info
			);
	hipDeviceSynchronize();
//	int info_gpu;
//	hipMemcpy(&info_gpu,d_info,sizeof(int),hipMemcpyDeviceToHost);

	hipMemcpy(res,d_req,sizeof(dt)*k*r,hipMemcpyDeviceToHost);
//	printTensor(res,k,r,1);
	hipFree(d_MM);

	dt *sum = new dt[r]();
	for(int i = 0;i<r;i++){
		for(int j = 0;j<k;j++){
				sum[i] += res[j*r+i]*res[j*r+i]; 
			}
		}

//	printTensor(sum,r,1,1);
	dt *d_sum;
	hipMalloc((void**)&d_sum,sizeof(dt)*r);
	hipMemcpy(d_sum,sum,sizeof(dt)*r,hipMemcpyHostToDevice);
	dim3 block4((k*r+512-1)/512,1,1);
	norm<<<block4,thread>>>(d_req,d_sum,k,r);
	hipMemcpy(res,d_req,sizeof(dt)*k*r,hipMemcpyDeviceToHost);
	hipFree(d_sum);
	delete[] sum;sum=nullptr;

	hipFree(d_req);
	hipFree(d_info);
	hipFree(d_work);
	hipsolverDnDestroy(handle1);
	hipDeviceReset();

}

void KRao(dt *X,dt *M,dt *N,dt *left,dt *right,int m,int n,int r,int k,int flag){
// m*r  n*r  m*n*r
	dt *d_M;
	hipMalloc((void **)&d_M,sizeof(dt)*m*r);
	hipMemcpy(d_M,M,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	
	dt *d_MT;
	hipMalloc((void **)&d_MT,sizeof(dt)*m*r);
	dim3 threads(512,1,1);
	dim3 blocks1((m*r+512-1)/512,1,1);
	transpose<<<blocks1,threads>>>(d_M,d_MT,m,r);
	half *d_m;
	half *d_mt;
	hipMalloc((void **)&d_m,sizeof(half)*m*r);
	hipMalloc((void **)&d_mt,sizeof(half)*m*r);
	f2h<<<blocks1,threads>>>(d_M,d_m,m*r);
	f2h<<<blocks1,threads>>>(d_MT,d_mt,m*r);
	hipFree(d_MT);

	dt *d_MTM;
	hipMalloc((void **)&d_MTM,sizeof(dt)*r*r);

	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r,
			r,
			m,
			&alpha,
			d_m,
			HIP_R_16F,
			r,
			d_mt,
			HIP_R_16F,
			m,
			&beta,
			d_MTM,
			HIP_R_32F,
			r,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	hipFree(d_m);
	hipFree(d_mt);



	dt *d_N;
	dt *d_NT;
	hipMalloc((void **)&d_N,sizeof(dt)*n*r);
	hipMemcpy(d_N,N,sizeof(dt)*n*r,hipMemcpyHostToDevice);

	hipMalloc((void **)&d_NT,sizeof(dt)*n*r);
	dim3 blocks2((n*r+512-1)/512,1,1);
	transpose<<<blocks2,threads>>>(d_N,d_NT,n,r);
	half *d_n;
	half *d_nt;
	hipMalloc((void **)&d_n,sizeof(half)*n*r);
	hipMalloc((void **)&d_nt,sizeof(half)*n*r);
	
	f2h<<<blocks2,threads>>>(d_N,d_n,n*r);
	f2h<<<blocks2,threads>>>(d_NT,d_nt,n*r);
	hipFree(d_NT);

	//now d_MT*M  d_NT*N

	dt *d_NTN;
	hipMalloc((void**)&d_NTN,sizeof(dt)*r*r);
	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r,
			r,
			n,
			&alpha,
			d_n,
			HIP_R_16F,
			r,
			d_nt,
			HIP_R_16F,
			n,
			&beta,
			d_NTN,
			HIP_R_32F,
			r,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	hipFree(d_n);
	hipFree(d_nt);

	dim3 blocks3((r*r+512-1)/512,1,1);
	elepro<<<blocks3,threads>>>(d_MTM,d_NTN,r);
	hipMemcpy(right,d_MTM,sizeof(dt)*r*r,hipMemcpyDeviceToHost);
	hipFree(d_MTM);
	hipFree(d_NTN);

	//right is solve the right

	dt *d_dot;
	hipMalloc((void **)&d_dot,sizeof(dt)*m*n*r);
	dim3 blocks((m*n*r+512-1)/512,1,1);
	krpro<<<blocks,threads>>>(d_M,d_N,d_dot,m,n,r);
	hipFree(d_M);
	hipFree(d_N);
	//res store the dotpro  bc*a

	dt *d_X;
	dt *d_X_M;
	hipMalloc((void**)&d_X,sizeof(dt)*m*n*k);
	hipMalloc((void**)&d_X_M,sizeof(dt)*m*n*k);
	hipMemcpy(d_X,X,sizeof(dt)*m*n*k,hipMemcpyHostToDevice);

	dim3 blocks4((m*n*k+512-1)/512,1,1);
	if(flag == 1){
		mode1tran<<<blocks4,threads>>>(d_X,d_X_M,k,n,m);
	}else if(flag == 2){
		mode2tran<<<blocks4,threads>>>(d_X,d_X_M,n,k,m);
	}else{
		mode3tran<<<blocks4,threads>>>(d_X,d_X_M,n,m,k);
	}
	hipFree(d_X);

	half *d_x_m;
	half *d_hdot;
	hipMalloc((void**)&d_x_m,sizeof(half)*m*n*k);
	hipMalloc((void**)&d_hdot,sizeof(half)*m*n*r);
	f2h<<<blocks4,threads>>>(d_X_M,d_x_m,m*n*k);
	f2h<<<blocks,threads>>>(d_dot,d_hdot,m*n*r);

	hipFree(d_X_M);
	hipFree(d_dot);

	// d_X1*d_dot = left
	dt *d_left;
	hipMalloc((void**)&d_left,sizeof(dt)*k*r);
	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r,
			k,
			m*n,
			&alpha,
			d_hdot,
			HIP_R_16F,
			r,
			d_x_m,
			HIP_R_16F,
			m*n,
			&beta,
			d_left,
			HIP_R_32F,
			r,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	hipMemcpy(left,d_left,sizeof(dt)*k*r,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_x_m);
	hipFree(d_hdot);
	hipFree(d_left);

}



void solve(dt *left,dt *right,dt *res,int r,int m){
	dt *d_left;
	dt *d_right;
	hipMalloc((void**)&d_right,sizeof(dt)*m*r);
	hipMalloc((void**)&d_left,sizeof(dt)*r*r);
	dt *d_work;
	int *d_info;
	int lwork;
	hipsolverHandle_t handle;
	hipsolverDnCreate(&handle);
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipMalloc((void**)&d_info,sizeof(int));

	hipMemcpy(d_left,left,sizeof(dt)*r*r,hipMemcpyHostToDevice);
	hipMemcpy(d_right,right,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	hipsolverDnSpotrf_bufferSize(
			handle,
			uplo,
			r,
			d_left,
			r,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);

	hipsolverDnSpotrf(
			handle,
			uplo,
			r,
			d_left,
			r,
			d_work,
			lwork,
			d_info
			);
	hipsolverDnSpotrs(
			handle,
			uplo,
			r,
			m,
			d_left,
			r,
			d_right,
			r,
			d_info
			);
	hipDeviceSynchronize();
//	int info_gpu;
//	hipMemcpy(&info_gpu,d_info,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(res,d_right,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
/*	if(info_gpu == 0){
		cout<<"OK"<<endl;
		cout<<endl;
	}
*/
// d_right store the A/B/C m*r


//	printTensor(res,m,r,1);
	dt *sum = new dt[r]();
	for(int i = 0;i<r;i++){
		for(int j = 0;j<m;j++){
				sum[i] += res[j*r+i]*res[j*r+i]; 
			}
		}

//	printTensor(sum,r,1,1);
	dt *d_sum;
	hipMalloc((void**)&d_sum,sizeof(dt)*r);
	hipMemcpy(d_sum,sum,sizeof(dt)*r,hipMemcpyHostToDevice);
	dim3 threads(512,1,1);
	dim3 blocks((m*r+512-1)/512,1,1);
	norm<<<blocks,threads>>>(d_right,d_sum,m,r);
	hipMemcpy(res,d_right,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
	hipFree(d_sum);
	hipFree(d_left);
	hipFree(d_right);
	hipFree(d_info);
	hipFree(d_work);
	hipsolverDnDestroy(handle);
	hipDeviceReset();
	delete[] sum;sum = nullptr;


}
void cp_als(dt *X,dt *A,dt *B,dt *C,int a,int b,int c,int r){

/*	dt *X_temp = new dt[a*b*c]();
	dt *error = new dt[100];
	dt tol = 1e06;
	dt X_norm = 0.0;
	for(int i = 0;i<a*b*c;i++){
		X_norm += X[i]*X[i];
	}
	X_norm = sqrt(X_norm);
*/
    clock_t lu,guo,be,yi;
    lu =clock();
	dt *temp1 = new dt[a*r]();
	dt *temp2 = new dt[b*r]();
	dt *temp3 = new dt[c*r]();
	dt *tem1 = new dt[r*r]();
	dt *tem2 = new dt[r*r]();
	dt *tem3 = new dt[r*r]();
	guo = clock();
	cout<<"分配空间"<<"  ";
	cout<<(double)(guo-lu)/CLOCKS_PER_SEC<<"s"<<endl;
	
	for(int i = 0;i<1;i++){
        
        clock_t t1,t2,t3;
        t1 = clock();
		KRao(X,C,B,temp1,tem1,c,b,r,a,1);
        t2 = clock();
    cout<<"计算左右值"<<"  "; 
	cout<<(double)(t2-t1)/CLOCKS_PER_SEC<<"s"<<endl;
      
		solve(tem1,temp1,A,r,a);     // we get A  
    t3 = clock();
    cout<<"zuizhongzhi"<<"  "; 
	cout<<(double)(t3-t2)/CLOCKS_PER_SEC<<"s"<<endl;
    

		KRao(X,C,A,temp2,tem2,c,a,r,b,2);
		solve(tem2,temp2,B,r,b);     // we get B
		
		KRao(X,B,A,temp3,tem3,b,a,r,c,3);
		solve(tem3,temp3,C,r,c);    //we get C

//		recontr(X,X_temp,A,B,C,a,b,c,r);

//		error[i] = 
	}
    yi = clock();
	delete[] temp1;temp1 = nullptr;
	delete[] temp2;temp1 = nullptr;
	delete[] temp3;temp1 = nullptr;
	delete[] tem1;tem1 = nullptr;
	delete[] tem2;tem2 = nullptr;
	delete[] tem3;tem3 = nullptr;
	be = clock();
	cout<<"释放空间"<<"  ";
	cout<<(double)(be-yi)/CLOCKS_PER_SEC<<"s"<<endl;
}

/*dt norm(dt *X,int a,int b,int c){
	dt temp = 0.0;
	dt *d_X;
	dt d_temp;
	hipMalloc(d_X,sizeof(dt)*a*b*c);
	hipMalloc(d_temp,sizeof(dt));
	hipMemcpy(d_X,X,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	dim3 threads(512,1,1);
	dim3 blocks((a*b*c+512-1)/512,1,1);
	Norm<<<blocks,threads>>>(d_X,d_temp,a,b,c);
	hipMemcpy(temp,d_temp)

	
}*/

/*void recontr(dt *src,dt *des,dt *A,dt *B,dt *C,int a,int b,int c,int r){
	dt *d_C;
	dt *d_B;
	dt *d_A;
	dt *d_des;
	dt *d_src;
	dt *d_BA;

	hipMalloc(d_B,sizeof(dt)*b*r);
	hipMalloc(d_A,sizeof(dt)*a*r);
	hipMalloc(d_CB,sizeof(dt)*a*b*r);
	hipMemcpy(d_B,B,sizeof(dt)*b*r);
	*/
