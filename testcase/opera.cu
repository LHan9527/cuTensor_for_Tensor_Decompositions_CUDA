#include "head.h"

void Btensor2mat(dt *A,dt *A1,dt *A2,dt *A3,int a,int b,int c){
	
	dt *d_AA;
	dt *d_A1;
	dt *d_A2;
	dt *d_A3;

	hipMalloc((void **)&d_AA,sizeof(dt)*a*b*c);
	hipMalloc((void **)&d_A1,sizeof(dt)*a*b*c);

	hipMemcpy(d_AA,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);

	dim3 threads(512,1,1);
	dim3 blocks(((a*b*c+512-1)/512),1,1);

	mode1tran<<<blocks,threads>>>(d_AA,d_A1,a,b,c);
	hipMemcpy(A1,d_A1,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);
	hipFree(d_A1);

	hipMalloc((void **)&d_A2,sizeof(dt)*a*b*c);
	mode2tran<<<blocks,threads>>>(d_AA,d_A2,a,b,c);
	hipMemcpy(A2,d_A2,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);
	hipFree(d_A2);

	hipMalloc((void **)&d_A3,sizeof(dt)*a*b*c);
	mode3tran<<<blocks,threads>>>(d_AA,d_A3,a,b,c);

	hipMemcpy(A3,d_A3,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);

	hipFree(d_AA);
	hipFree(d_A3);

	cout<<"Btensor2mat is over"<<endl;

/*		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int j = 0;j<b;j++){
					t2m[i*b*c+k*b+j] = A[k*a*b+i*b+j];
				}
			}
		}
	
		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int j = 0;j<b;j++){
					t2m[j*a*c+k*a+i] = A[k*a*b+i*b+j];
				}
			}
		}

		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int  j = 0;j<b;j++){
					 t2m[k*a*b+j*a+i]= A[k*a*b+i*b+j];
				}
			}
		}

*/

}

void getvector1(dt *A,dt *U,int m,int n,int r){
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*m*n);
	hipMemcpy(d_A,A,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	dt *d_C;
	hipMalloc((void**)&d_C,sizeof(dt)*m*m);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipblasSsyrk(
		handle,
		uplo,
		HIPBLAS_OP_T,
		m,n,
		&alpha,
		d_A,n,
		&beta,
		d_C,m	
		);

	hipsolverHandle_t cusolverH = NULL;
	dt *d_V;
	hipMalloc((void**)&d_V,sizeof(dt)*m*r);
	dt *d_W;
	int *devInfo = NULL;
	dt *d_work = NULL;
	int lwork;
	int info_gpu = 0;
	hipsolverDnCreate(&cusolverH);
	hipMalloc((void**)&devInfo,sizeof(int));
	hipMalloc((void**)&d_W,sizeof(dt)*m);
	
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipsolverDnSsyevd_bufferSize(
			cusolverH,
			jobz,
			uplo,
			m,
			d_C,
			m,
			d_W,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);

	hipsolverDnSsyevd(
			cusolverH,
			jobz,
			uplo,
			m,
			d_C,   //store vectors
			m,
			d_W,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
/*	dt *hh = new dt[m*r]();
	hipMemcpy(hh,d_C+m*(m-r),sizeof(dt)*m*r,hipMemcpyDeviceToHost);
	printTensor(hh,m,r,1);
	delete[] hh,hh=nullptr;
	
	dt *Ctemp;
	hipMalloc((void**)&Ctemp,sizeof(dt)*m*r);
	Ctemp = d_C+m*(m-r);
	*/
//	hipMemcpy(V,d_C,sizeof(dt)*m*m,hipMemcpyDeviceToHost);
	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
	if(info_gpu == 0){
		cout<<"ok"<<endl;
	}else{
		cout<<info_gpu<<endl;
	}
	// now V is vectors 
	hipFree(d_W);
	hipFree(d_A);
	hipFree(d_work);
	hipFree(devInfo);
	hipsolverDnDestroy(cusolverH);
	
	hipblasSgeam(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		r,m,
		&alpha,
		d_C+m*(m-r),
//		Ctemp,
		m,
		&beta,
		d_V,
		r,
		d_V,r
		);
	hipMemcpy(U,d_V,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
//	printTensor(U,m,r,1);

	hipFree(d_C);
	//printTensor(W,m,1,1);
	hipFree(d_V);
	hipblasDestroy(handle);
	hipDeviceReset();

}

void getvector(dt *A,dt *U,int m,int n,int r){
	//we compute A*A'
	dt *d_A;
	dt *d_AT;
	hipMalloc((void**)&d_A,sizeof(dt)*m*n);
	hipMalloc((void**)&d_AT,sizeof(dt)*m*n);
	dt *d_AAT;
	hipMalloc((void**)&d_AAT,sizeof(dt)*m*m);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipMemcpy(d_A,A,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	dim3 threads(512,1,1);
	dim3 blocks((m*n+512-1)/512,1,1);
	transpose<<<blocks,threads>>>(d_A,d_AT,m,n);  // now d_AT n*m
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			m,
			m,
			n,
			&alpha,
			d_AT,
			m,
			d_A,
			n,
			&beta,
			d_AAT,  //store A*A'
			m
			);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_AT);
// eig
	hipsolverHandle_t cusolverH = NULL;
	dt *V = new dt[m*m]();
	dt *V1 = new dt[r*m]();
	dt *d_W;
	int *devInfo = NULL;
	dt *d_work = NULL;
	int lwork;
	int info_gpu = 0;
	hipsolverDnCreate(&cusolverH);
	hipMalloc((void**)&devInfo,sizeof(int));
	hipMalloc((void**)&d_W,sizeof(dt)*m);
	
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverDnSsyevd_bufferSize(
			cusolverH,
			jobz,
			uplo,
			m,
			d_AAT,
			m,
			d_W,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);

	hipsolverDnSsyevd(
			cusolverH,
			jobz,
			uplo,
			m,
			d_AAT,   //store vectors
			m,
			d_W,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
	hipMemcpy(V,d_AAT,sizeof(dt)*m*m,hipMemcpyDeviceToHost);
//	hipMemcpy(W,d_W,sizeof(dt)*m,hipMemcpyDeviceToHost);
	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
	if(info_gpu == 0){
		cout<<"ok"<<endl;
	}else{
		cout<<info_gpu<<endl;
	}

	hipFree(d_W);
	hipFree(d_work);
	hipFree(devInfo);
	hipFree(d_AAT);
//	printTensor(V,m,m,1);
//	printTensor(W,m,1,1);
	hipsolverDnDestroy(cusolverH);
	hipDeviceReset();
//	printTensor(V,m,m,1);
	for(int i=0;i<r;i++){
		for(int j = 0;j<m;j++){
			V1[i*m+j] = V[i*m+j+m*(m-r)];
			U[j*r+i] = V1[i*m+j];
		}
	}
//	printTensor(U,m,r,1);
		
	delete[] V;V=nullptr;
	delete[] V1;V1=nullptr;

}






