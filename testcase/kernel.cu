#include "hip/hip_runtime.h"
#include "head.h"

__global__ void transpose(dt *A,dt* AT,int m,int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<n*m){
		int row = i/n;
		int col = i%n;
		AT[col*m+row] = A[row*n+col];
		i+=temp;
	}
    __syncthreads();
}

__global__ void mode1tran(dt *AA,dt *A1,int a,int b,int c){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<a*b*c){
		int tube = i/(a*b);		//which slice	
		int row = (i-tube*(a*b))/b;	
		int col = (i-tube*(a*b))%b;		//get the index
		A1[row*b*c+tube*b+col] = AA[tube*a*b+row*b+col];
		i+=temp;
	}
    __syncthreads();
}

__global__ void mode2tran(dt *AA,dt *A2,int a,int b,int c){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<a*b*c){
		int tube = i/(a*b);		//which slice	
		int row = (i-tube*(a*b))/b;	
		int col = (i-tube*(a*b))%b;		//get the index
		A2[col*a*c+tube*a+row] = AA[tube*a*b+row*b+col];
		i+=temp;
	}
    __syncthreads();
}

__global__ void mode3tran(dt *AA,dt *A3,int a,int b,int c){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<a*b*c){
		int tube = i/(a*b);		//which slice	
		int row = (i-tube*(a*b))/b;	
		int col = (i-tube*(a*b))%b;		//get the index
		A3[tube*a*b+col*a+row] = AA[tube*a*b+row*b+col];
		i+=temp;
	}
    __syncthreads();
}
__global__ void tran3mode(dt *AA,dt *A3,int a,int b,int c){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<a*b*c){
		int tube = i/(a*b);		//which slice	
		int row = (i-tube*(a*b))/b;	
		int col = (i-tube*(a*b))%b;		//get the index
		A3[tube*a*b+row*b+col] = AA[tube*a*b+col*a+row];
		i+=temp;
	}
    __syncthreads();
}

__global__ void krpro(dt *M,dt *N,dt *res,int m,int n,int r){
	
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<m*n*r){
		int row = i/r;
		int col = i%r;
		res[row*r+col] = M[(row/n)*r+col]*N[(row%n)*r+col];
		i+=temp;
	}
    __syncthreads();
}

__global__ void elepro(dt *A,dt *B,int r){
	
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<r*r){
		A[i] = A[i]*B[i];
		i+=temp;
	}
    __syncthreads();
}

__global__ void norm(dt *A,dt *B,int m ,int r){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<m*r){
		int row = i/r;
		int col = i%r;
		A[row*r+col] = A[row*r+col]/sqrt(B[col]);
		i+=temp;
	}
    __syncthreads();
}

__global__ void kronpro(dt *M,dt *N,dt *res,int m,int n,int k,int r ){
	// M is m*n   N is k*r  res is mk*nr
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<m*n*k*r){
		int row = i/(n*r);
		int col = i%(n*r);
		res[row*(n*r)+col] = M[(row/k)*n+(col/r)]*N[(row%k)*r+(col%r)];
		i+=temp;
	}
    __syncthreads();
}

__global__ void matvec(dt *M,dt *N,int m){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<m*m){
		int row = i/m;
		int col = i%m;
		if(N[row]>1e-06){
			M[row*m+col] = M[row*m+col]*(1/N[row]);
		}else{
			M[row*m+col] = 0.0;
		}
		i+=temp;
	}
    __syncthreads();
}

__global__ void f2h(dt *A,half *B,int num){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<num){
		B[i] = __float2half(A[i]);
		i+=temp;
	}
	__syncthreads();
}
