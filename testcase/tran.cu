#include "hip/hip_runtime.h"
#include "head.h"
using namespace std;

int main(int argc,char *argv[]){
	int m,n,k;
//	int a = atoi(argv[1]);
//	int b = atoi(argv[2]);
//	int c = atoi(argv[3]);
	int cnt[10] = {80,128,256,512,640,704,768,832,896,960};
for(int i = 0;i<10;i++){
//	int m = 2; 
//	int n = 2; 
//	int k = 2; 
	int a = cnt[i];
	int b = a;
	int c = a;
	if(a<10){
		m = 1;
		 n=1;
		 k=1;
	}else{
	 m = a/8;
	 n = b/8;
	 k = c/8;}
	dt *A = new dt[a*b*c]();
	dt *res1 = new dt[m*n*k]();
	dt *U1 = new dt[a*m]();
	dt *U2 = new dt[a*m]();
	dt *U3 = new dt[c*k]();

	srand(0);
	for(int i = 0;i<a*b*c;i++){
		A[i] = rand()/RAND_MAX;
	//	A[i] = rand()%4;
	}

	for(int i = 0;i<a*m;i++){
	//	U1[i] = rand()%4;
		U1[i] = rand()/(float)RAND_MAX;
	}
	for(int i = 0;i<b*n;i++){
	//	U2[i] = rand()%4;
		U2[i] = rand()/(float)RAND_MAX;
	}
	for(int i = 0;i<c*k;i++){
	//	U3[i] = rand()%4;
		U3[i] = rand()/(float)RAND_MAX;
	}

//	printTensor(A,a,b,c);


//	printTensor(U1,a,m,1);
//	printTensor(U2,b,n,1);
//	printTensor(U3,c,k,1);
	Hosvd(A,res1,U1,U2,U3,a,b,c);
	Hosvd(A,res1,U1,U2,U3,a,b,c);
	double sum = 0.0;
for(int j = 0;j<5;j++){
	if(a<10){
		 m = 1;
		 n=1;
		 k=1;
	}else{
	 m = a/8;
	 n = b/8;
	 k = c/8;}

	clock_t tt1,tt2;
	tt1=clock();
	dt *A1 = new dt[a*b*c]();
	dt *A2 = new dt[a*b*c]();
	dt *A3 = new dt[a*b*c]();
	Btensor2mat(A,A1,A2,A3,a,b,c);
	getvector1(A1,U1,a,b*c,m);
	getvector1(A2,U2,b,a*c,n);
	getvector1(A3,U3,c,b*a,k);
	newtest16(A,U1,U2,U3,res1,a,b,c);
	
	delete[] A1;  A1=nullptr;
	delete[] A2;  A2=nullptr;
	delete[] A3;  A3=nullptr;
	tt2=clock();
//	cout<<(double)(t2-t1)/CLOCKS_PER_SEC<<"s"<<endl;
	sum = sum+(double)(tt2-tt1)/CLOCKS_PER_SEC;
}
	ofstream outfile("ttime.txt",ios::app);
	outfile<<a<<"*"<<a<<"*"<<a<<"  ";
	outfile<<sum/5<<"s"<<endl;
	outfile.close();
	cout<<"all is over"<<endl;

//	printTensor(A1,a,b*c,1);
//	printTensor(A2,b,a*c,1);
//	printTensor(A3,c,b*a,1);

//	getvector1(A1,U1,a,b*c,m);
//	getvector1(A2,U2,b,a*c,n);
//	getvector1(A3,U3,c,b*a,k);

//	newtest16(A,U1,U2,U3,res1,a,b,c);
/*	printTensor(U1,a,2,1);
*/
//	time4 = clock();
//	printTensor(U2,a,2,1);

//	newtest(A,U1,U2,U3,res1,a,b,c);


	
//	cuStrideMode(A,U1,res2,a,b,c);
//	time3 = clock();

/*	newtest16(A,U1,U2,U3,res2,a,b,c);
	time3 = clock();
	newtest16h(A,U1,U2,U3,res3,a,b,c);
*/
//	cout<<(double)(time3-time2)/CLOCKS_PER_SEC<<"s"<<endl;
//	cout<<(double)(time4-time3)/CLOCKS_PER_SEC<<"s"<<endl;

//	printTensor(res1,m,n,k);
//	printTensor(res2,m,b,c);

	delete[] A;  A=nullptr;
	delete[] U1;  U1=nullptr;
	delete[] U2;  U2=nullptr;
	delete[] U3;  U3=nullptr;
	delete[] res1; res1=nullptr;
}
	return 0;

}
