#include "hip/hip_runtime.h"
#include "head.h"
using namespace std;

int main(int argc,char *argv[]){
	int a = atoi(argv[1]);
	int b = atoi(argv[2]);
	int c = atoi(argv[3]);
	dt *A = new dt[a*b*c]();
//	int m = 2; 
//	int n = 2; 
//	int k = 2; 
	int m = 0.1*a;
	int n = 0.1*b;
	int k = 0.1*c;
	dt *res1 = new dt[m*n*k]();
	dt *res2 = new dt[m*n*k]();
	dt *res3 = new dt[m*n*k]();
	dt *U1 = new dt[a*m]();
	dt *U2 = new dt[a*m]();
	dt *U3 = new dt[c*k]();

	srand(2);
	for(int i = 0;i<a*b*c;i++){
		A[i] = rand()/RAND_MAX;
	//	A[i] = rand()%4;
	}

	for(int i = 0;i<a*m;i++){
	//	U1[i] = rand()%4;
		U1[i] = rand()/(float)RAND_MAX;
	}
	for(int i = 0;i<b*n;i++){
	//	U2[i] = rand()%4;
		U2[i] = rand()/(float)RAND_MAX;
	}
	for(int i = 0;i<c*k;i++){
	//	U3[i] = rand()%4;
		U3[i] = rand()/(float)RAND_MAX;
	}

//	printTensor(A,a,b,c);
	dt *A1 = new dt[a*b*c]();
	dt *A2 = new dt[a*b*c]();
	dt *A3 = new dt[a*b*c]();


//	printTensor(U1,a,m,1);
//	printTensor(U2,b,n,1);
//	printTensor(U3,c,k,1);
	clock_t time1,time2,time3,time4;

	time1 = clock();
	Btensor2mat(A,A1,A2,A3,a,b,c);
//	printTensor(A1,a,b*c,1);
//	printTensor(A2,b,a*c,1);
//	printTensor(A3,c,b*a,1);

	getvector1(A1,U1,a,b*c,m);
	getvector1(A2,U2,b,a*c,n);
	getvector1(A3,U3,c,b*a,k);

	newtest16(A,U1,U2,U3,res1,a,b,c);
	time2 = clock();

/*	printTensor(U1,a,2,1);
	getvector(A1,U1,a,b*c,m);
	getvector(A2,U2,b,a*c,n);
	getvector(A3,U3,c,b*a,k);
*/
//	time4 = clock();
//	printTensor(U2,a,2,1);

//	newtest(A,U1,U2,U3,res1,a,b,c);


	
//	cuStrideMode(A,U1,res2,a,b,c);
//	time3 = clock();

/*	newtest16(A,U1,U2,U3,res2,a,b,c);
	time3 = clock();
	newtest16h(A,U1,U2,U3,res3,a,b,c);
*/
	cout<<(double)(time2-time1)/CLOCKS_PER_SEC<<"s"<<endl;
//	cout<<(double)(time3-time2)/CLOCKS_PER_SEC<<"s"<<endl;
//	cout<<(double)(time4-time3)/CLOCKS_PER_SEC<<"s"<<endl;

//	printTensor(res1,m,n,k);
//	printTensor(res2,m,b,c);

	delete[] A;  A=nullptr;
	delete[] A1;  A1=nullptr;
	delete[] A2;  A2=nullptr;
	delete[] A3;  A3=nullptr;
	delete[] U1;  U1=nullptr;
	delete[] U2;  U2=nullptr;
	delete[] U3;  U3=nullptr;
	delete[] res1; res1=nullptr;
	delete[] res2; res2=nullptr;
	delete[] res3; res3=nullptr;

	return 0;

}
