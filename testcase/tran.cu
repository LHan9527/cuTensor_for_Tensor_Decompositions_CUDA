#include "hip/hip_runtime.h"
#include "head.h"
using namespace std;

int main(int argc,char *argv[]){
	int a = atoi(argv[1]);
	int b = atoi(argv[2]);
	int c = atoi(argv[3]);
	//A a*b  B a*c  res b*c
	dt *A = new dt[a*b*c]();
	dt *B = new dt[a*b]();
	dt *res = new dt[a*a*c]();
	dt *res2 = new dt[a*a*c]();

	for(int i = 0;i<a*b*c;i++){
		A[i] = rand()*0.1/(RAND_MAX*0.1);
	}
//	printTensor(A,a,b,1);
	for(int i = 0;i<a*b;i++){
		B[i] = rand()*0.1/(RAND_MAX*0.1);
	}
//	printTensor(B,a,b,1);

	clock_t time1,time2,time3;
	time1 = clock();

	cuStrideModetran(A,B,res, a, b, c);
	cout<<endl;
	time2 = clock();
	cuStrideModenotran(A,B,res2, a, b, c);
	time3 = clock();

	cout<<(double)(time2-time1)/CLOCKS_PER_SEC<<"s"<<endl;
	cout<<(double)(time3-time2)/CLOCKS_PER_SEC<<"s"<<endl;
//	printTensor(res,b,c,1);
//	printTensor(res2,b,c,1);

	delete[] A;  A=nullptr;
	delete[] B; B=nullptr;
	delete[] res;  res=nullptr;
	delete[] res2;  res2=nullptr;

	return 0;

}
