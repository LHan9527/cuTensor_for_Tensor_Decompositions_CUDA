#include "hip/hip_runtime.h"
#include "head.h"
using namespace std;

int main(int argc,char *argv[]){
	int r;
	int cnt[10] = {80,128,256,512,640,704,768,832,896,960};
for(int i = 0;i<10;i++){
	int a = cnt[i];
	int b = a;
	int c = a;
	if(a<10){
		r = 1;
	}else{
	 r = a/8;
	}
	dt *X = new dt[a*b*c]();
	srand(0);
	for(int i = 0;i<a*b*c;i++){
		X[i] = rand()*0.1/(RAND_MAX*0.1);
	}
//	printTensor(X,a,b,c);
	dt *A = new dt[a*r]();
	dt *B = new dt[b*r]();
	dt *C = new dt[c*r]();
	for(int i = 0;i<a*r;i++){
		A[i] = rand()*0.1/(RAND_MAX*0.1);
	}
//	printTensor(A,a,r,1);
	for(int i = 0;i<b*r;i++){
		B[i] = rand()*0.1/(RAND_MAX*0.1);
	}
//	printTensor(B,b,r,1);
	for(int i = 0;i<c*r;i++){
		C[i] = rand()*0.1/(RAND_MAX*0.1);
	}
//	printTensor(C,c,r,1);
	cp_als(X,A,B,C,a,b,c,r);
	cp_als(X,A,B,C,a,b,c,r);
	double sum = 0.0;
for(int j = 0;j<5;j++){
/*	dt *temp1 = new dt[a*r]();
	dt *temp2 = new dt[b*r]();
	dt *temp3 = new dt[c*r]();
	dt *tem1 = new dt[r*r]();
	dt *tem2 = new dt[r*r]();
	dt *tem3 = new dt[r*r]();
*/
	clock_t start,end;
	start = clock();
	for(int mei = 0;mei<1;mei++){

		allin(X,C,B,A,c,b,r,a,1);
	//	KRao(X,C,B,temp1,tem1,c,b,r,a,1);
	//	solve(tem1,temp1,A,r,a);     // we get A  

		allin(X,C,A,B,c,a,r,b,2);
	//	KRao(X,C,A,temp2,tem2,c,a,r,b,2);
	//	solve(tem2,temp2,B,r,b);     // we get B
		
		allin(X,B,A,C,b,a,r,c,3);
	//	KRao(X,B,A,temp3,tem3,b,a,r,c,3);
	//	solve(tem3,temp3,C,r,c);    //we get C

//		recontr(X,X_temp,A,B,C,a,b,c,r);

//		error[i] = 
	}
	end=clock();
	sum = sum+(double)(end-start)/CLOCKS_PER_SEC;
/*	delete[] temp1;temp1 = nullptr;
	delete[] temp2;temp1 = nullptr;
	delete[] temp3;temp1 = nullptr;
	delete[] tem1;tem1 = nullptr;
	delete[] tem2;tem2 = nullptr;
	delete[] tem3;tem3 = nullptr;
*/
	}


	ofstream outfl("ctime.txt",ios::app);
	outfl<<a<<"*"<<a<<"*"<<a<<"  ";
	outfl<<sum/5<<"s"<<endl;
	outfl.close();
	delete[] X;X = nullptr;
	delete[] A;A = nullptr;
	delete[] B;B = nullptr;
	delete[] C;C = nullptr;
//	break;
}
	return 0;

}
