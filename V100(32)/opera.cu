#include "hip/hip_runtime.h"

#include <stdlib.h>
#include "opera.h"
void SBgemm(dt *A,dt *core,dt *U1,dt *U2,dt *U3,int a,int b,int c){
	int r1 = a/8;
	int r2 = b/8;
	int r3 = c/8;
	while(r1%8!=0){
		r1--;
	}
	while(r2%8!=0){
		r2--;
	}
	while(r3%8!=0){
		r3--;
	}
cout<<"come in SBgemm"<<endl;
//	int r1 = 2;
//	int r2 = 3;
//	int r3 = 2;
//	dt *temp1 = new dt[b*r1*c]();
//	dt *temp2 = new dt[r1*r2*c]();
//	dt *temp3 = new dt[c*r1*r2]();	//mode3 mat
	//dt *temp = new dt[r3*r1*r2]();	//result to be convert

	//compute A ×1 U1'×2 U2'×3 U3'
	// first compute U1'[X1,X2,X3～～Xc]U2 = temp;   then temp*U3'   
	//  U1 a*r1  U1' r1*a
	// A a*b*c
	// U2 b*r2  U2' r2*b
	// U3 c*r3  U3' r3*c

	dt alpha = 1.0;
	dt beta = 0.0;
	dt *d_A;
	dt *d_U1;
	dt *d_U2;
	dt *d_U3;
	dt *d_temp1;
	dt *d_temp2;
	dt *d_temp3;
	dt *d_temp;
	hipMalloc((void **)&d_A,a*b*c*sizeof(dt));
	hipMalloc((void **)&d_U1,a*r1*sizeof(dt));
	hipMalloc((void **)&d_temp1,sizeof(dt)*b*r1*c);

	hipMemcpy(d_A,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipMemcpy(d_U1,U1,sizeof(dt)*a*r1,hipMemcpyHostToDevice);



	//hipMemcpy(d_C,C,sizeof(dt)*a*d*c);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	
	dim3 threads(512,1,1);
	

	hipblasGemmStridedBatchedEx(
			
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			b,				//row of A C
			r1,				//col of B C
			a,				//col of A ,row of B
			&alpha,
			d_A,
			HIP_R_32F,
			b,				//leading dimension store A
			b*a,			//step between two mat
		   	d_U1,
			HIP_R_32F,
			r1,
			0,
			&beta,
			d_temp1,
			HIP_R_32F,
			b,
			b*r1,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT

			);
	hipFree(d_U1);
	hipFree(d_A);
	//now d_temp1 store the real value col first
//	hipMemcpy(temp1,d_temp1,sizeof(dt)*b*r1*c,hipMemcpyDeviceToHost);
//	printTensor(temp1,r1,b,c);

	hipMalloc((void **)&d_U2,b*r2*sizeof(dt));
	hipMalloc((void **)&d_temp2,sizeof(dt)*r1*r2*c);
	hipMemcpy(d_U2,U2,sizeof(dt)*b*r2,hipMemcpyHostToDevice);

	

	hipblasGemmStridedBatchedEx(
			
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r2,
			r1,
			b,
			&alpha,
			d_U2,
			HIP_R_32F,
			r2,
			0,
			d_temp1,
			HIP_R_32F,
			b,
			b*r1,
			&beta,
			d_temp2,
			HIP_R_32F,
			r2,
			r2*r1,
			c,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT

			);

//	hipMemcpy(temp2,d_temp2,sizeof(dt)*r1*r2*c,hipMemcpyDeviceToHost);
	
//	printTensor(temp2,r1,r2,c);
	hipFree(d_U2);
	hipFree(d_temp1);

	hipMalloc((void **)&d_temp3,sizeof(dt)*r1*r2*c);	//mode 3 mat

	// now temp2 store the real value 
	//we will mat3,and the 
	dim3 blocks((r1*r2*c+512-1)/512,1,1);
	mode3tran<<<blocks,threads>>>(d_temp2,d_temp3,r1,r2,c);
//	temp3 = tensor2mat(temp2,r1,r2,c,3);
//	printTensor(temp3,c,r1*r2,1);
	
	hipFree(d_temp2);

//	hipMemcpy(d_temp3,temp3,sizeof(dt)*c*r1*r2,hipMemcpyHostToDevice);

	hipMalloc((void **)&d_U3,c*r3*sizeof(dt));
	hipMemcpy(d_U3,U3,sizeof(dt)*c*r3,hipMemcpyHostToDevice);
	hipMalloc((void **)&d_temp,sizeof(dt)*r1*r2*r3);

	

	hipblasGemmEx(
			
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			r1*r2,
			r3,
			c,
			&alpha,
			d_temp3,
			HIP_R_32F,
			r1*r2,
			d_U3,
			HIP_R_32F,
			r3,
			&beta,
			d_temp,
			HIP_R_32F,
			r1*r2,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT

			);

	hipFree(d_U3);
	hipFree(d_temp3);

	dim3 blocks1((r1*r2*r3+512-1)/512,1,1);
	dt *d_core;
	hipMalloc((void**)&d_core,sizeof(dt)*r1*r2*r3);
	tran3mode<<<blocks1,threads>>>(d_temp,d_core,r1,r2,r3);
	hipFree(d_temp);
	hipMemcpy(core,d_core,sizeof(dt)*r3*r1*r2,hipMemcpyDeviceToHost);

//	printTensor(temp,r3,r1*r2,1);
	hipFree(d_core);
	hipblasDestroy(handle);
	//mode3 to Tensor 
	//r3 * r1×r2   to r1*r2*r3
/*	for(int k = 0;k<r3;k++){
		for(int i = 0;i<r1;i++){
			for(int j = 0;j<r2;j++){
				core[k*r1*r2+i*r2+j] = temp[k*r1*r2+j*r1+i];
			}
		}
	}
*/

//	delete[] temp1; temp1 = nullptr;
//	delete[] temp2; temp2 = nullptr;
//	delete[] temp3; temp3 = nullptr;
//	delete[] temp; temp = nullptr;
	
	cout<<"SBgeem is over"<<endl;
}

void printTensor(dt *A,int a,int b,int c){
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<A[i*a*b+j*b+k]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
}

dt* tensor2mat(dt *A,int a,int b,int c,int mode){
	dt *t2m = new dt[a*b*c]();
	if (mode == 1){
		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int j = 0;j<b;j++){
					t2m[i*b*c+k*b+j] = A[k*a*b+i*b+j];
				}
			}
		}
		return t2m;
	}
	
	if (mode == 2){
		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int j = 0;j<b;j++){
					t2m[j*a*c+k*a+i] = A[k*a*b+i*b+j];
				}
			}
		}
		return t2m;
	}

	if (mode == 3){
		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int  j = 0;j<b;j++){
					 t2m[k*a*b+j*a+i]= A[k*a*b+i*b+j];
				}
			}
		}
		return t2m;
	}

	return t2m;
}

void msvd(dt *A,dt *U,int m,int n,int r){
	//printTensor(A,m,n,1);
	//printTensor(A,m,n,1);
	cout<<"come in svd"<<endl;
	dt* d_A;
	dt* d_AT;
	hipMalloc((void**)&d_AT,sizeof(dt)*m*n);
	hipMalloc((void**)&d_A,sizeof(dt)*m*n);
	hipMemcpy(d_A,A,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	dim3 threads(512,1,1);
	dim3 blocks(((m*n+512-1)/512),1,1);
	transpose<<<blocks,threads>>>(d_A,d_AT,m,n);	//mow d_A store the transpose of A
	//hipMemcpy(A,d_AT,sizeof(dt)*m*n,hipMemcpyDeviceToHost);
	// the we will use SVD  d_A col-store m*n
	dt* d_Utemp;			//left singular vectors
	hipMalloc((void**)&d_Utemp,sizeof(dt)*r*m);
	//dt* S = new dt[n];
	dt *d_U;
	dt *d_S;
	dt *d_V;
	int *d_info = NULL;
	int lwork = 0;
	dt *d_work = NULL;
	int info = 0;
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	const int econ = 0;

	hipsolverHandle_t cusolverH = NULL;
	hipsolverGesvdjInfo_t gesvdj_params = NULL;

	hipsolverDnCreate(&cusolverH);
	hipsolverDnCreateGesvdjInfo(&gesvdj_params);

	hipMalloc((void**)&d_U,sizeof(dt)*m*m);	
	hipMalloc((void**)&d_S,sizeof(dt)*n);	
	hipMalloc((void**)&d_V,sizeof(dt)*n*n);	
	hipMalloc((void**)&d_info,sizeof(int));
	hipsolverDnSgesvdj_bufferSize(
			cusolverH,
			jobz,
			econ,
			m,
			n,
			d_AT,
			m,
			d_S,
			d_U,
			m,
			d_V,
			n,
			&lwork,
			gesvdj_params
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgesvdj(
			cusolverH,
			jobz,
			econ,
			m,
			n,
			d_AT,
			m,
			d_S,
			d_U,
			m,
			d_V,
			n,
			d_work,
			lwork,
			d_info,
			gesvdj_params

			);
	hipDeviceSynchronize();
	//hipMemcpy(Utemp,d_U,sizeof(dt)*m*m,hipMemcpyDeviceToHost);
	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);
	if(0 == info){
		cout<<"ok"<<endl;
	}else if(0>info){
		cout<<-info<<"is wrong"<<endl;
	}else{
		cout<<info<<"do not work"<<endl;
	}

	dim3 thread1(512,1,1);
	dim3 block1(max(((m*n+512-1)/512),65535),1,1);
	transpose<<<block1,thread1>>>(d_U,d_Utemp,r,m);
	hipMemcpy(U,d_Utemp,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
//	printTensor(U,m,r,1);

	hipFree(d_A);
	hipFree(d_AT);
	hipFree(d_Utemp);
	hipFree(d_U);
	hipFree(d_S);
	hipFree(d_V);
	hipFree(d_work);
	hipFree(d_info);
	hipsolverDnDestroy(cusolverH);
	hipsolverDnDestroyGesvdjInfo(gesvdj_params);
	hipDeviceReset();

	
}

void HOSVD(dt *A,dt *core,dt *U1,dt *U2,dt *U3,int a,int b,int c){
	int r1 = a/8;
	int r2 = b/8;
	int r3 = c/8;
	while(r1%8!=0){
		r1--;
	}
	while(r2%8!=0){
		r2--;
	}
	while(r3%8!=0){
		r3--;
	}
//	int r1 = 2;
//	int r2 = 3;
//	int r3 = 2;
		
	dt *A1 = new dt[a*b*c]();	
	dt *A2 = new dt[a*b*c]();
	dt *A3 = new dt[a*b*c]();	//3 mode tensor to mat
	
/*	A1 = tensor2mat(A,a,b,c,1);		//a*bc
	A2 = tensor2mat(A,a,b,c,2);		//b*ac
	A3 = tensor2mat(A,a,b,c,3);		//c*ab  now we get 3 mode mats
*/
	Btensor2mat(A,A1,A2,A3,a,b,c);

//	msvd(A1,U1,a,b*c,r1);	//a*r1
//	msvd(A2,U2,b,a*c,r2);	//b*r2
//	msvd(A3,U3,c,a*b,r3);	//c*r3

	getvector(A1,U1,a,b*c,r1);
	getvector(A2,U2,b,a*c,r2);
	getvector(A3,U3,c,a*b,r3);
	//compute A ×1 U1'×2 U2'×3 U3'
	// first compute U1'[X1,X2,X3～～Xc]U2 = temp;   then temp*U3'   
	SBgemm(A,core,U1,U2,U3,a,b,c);
	


/*	cout<<"next to recover____________"<<endl;

	dt *rec = new dt[a*b*c]();		//store the recover Tensor	
	Recover(core,rec,U1,U2,U3,a,b,c);
	printTensor(rec,a,b,c);
	delete[] V;V=nullptr;
	delete[] rec; rec = nullptr;
*/

	delete[] A1; A1 = nullptr;
	delete[] A2; A2 = nullptr;
	delete[] A3; A3 = nullptr;

	}

// next is the function to return

void Recover(dt *core,dt *rec,dt *U1,dt *U2,dt *U3,int a ,int b,int c){
//	int r1 = 2;
//	int r2 = 3;
//	int r3 = 2;

	int r1 = 0.1*a;
	int r2 = 0.1*b;
	int r3 = 0.1*c;
//	dt *temp1 = new dt[r2*a*r3]();
	dt *temp2 = new dt[b*a*r3]();
	dt *temp3 = new dt[r3*a*b]();	//mode3 mat
	dt *temp = new dt[c*a*b]();	//result to be convert

	//compute A ×1 U1'×2 U2'×3 U3'
	// first compute U1'[X1,X2,X3～～Xc]U2 = temp;   then temp*U3'   
	//  U1 a*r1  U1' r1*a
	// A a*b*c
	// U2 b*r2  U2' r2*b
	// U3 c*r3  U3' r3*c

	dt alpha = 1.0;
	dt beta = 0.0;
	dt *d_core;
	dt *d_U1;
	dt *d_U2;
	dt *d_U3;
	dt *d_temp1;
	dt *d_temp2;
	dt *d_temp3;
	dt *d_temp;
	//
	hipMalloc((void **)&d_core,r1*r2*r3*sizeof(dt));
	hipMalloc((void **)&d_U1,a*r1*sizeof(dt));
	hipMalloc((void **)&d_U2,b*r2*sizeof(dt));
	hipMalloc((void **)&d_U3,c*r3*sizeof(dt));
	hipMalloc((void **)&d_temp1,sizeof(dt)*r2*a*r3);
	hipMalloc((void **)&d_temp2,sizeof(dt)*b*a*r3);
	hipMalloc((void **)&d_temp3,sizeof(dt)*r3*a*b);	//mode 3 mat

	hipMalloc((void **)&d_temp,sizeof(dt)*c*a*b);

	hipMemcpy(d_core,core,sizeof(dt)*r1*r2*r3,hipMemcpyHostToDevice);
	hipMemcpy(d_U1,U1,sizeof(dt)*a*r1,hipMemcpyHostToDevice);
	hipMemcpy(d_U2,U2,sizeof(dt)*b*r2,hipMemcpyHostToDevice);
	hipMemcpy(d_U3,U3,sizeof(dt)*c*r3,hipMemcpyHostToDevice);

	//hipMemcpy(d_C,C,sizeof(dt)*a*d*c);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSgemmStridedBatched(
			
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r2,				//row of A C
			a,				//col of B C
			r1,				//col of A ,row of B
			&alpha,
			d_core,
			r2,				//leading dimension store A
			r1*r2,			//step between two mat
		    d_U1,
			r1,
			0,
			&beta,
			d_temp1,
			r2,
			a*r2,
			r3				//batch number
			);
	//now d_temp1 store the real value col first
	
//	hipMemcpy(temp1,d_temp1,sizeof(dt)*a*r2*r3,hipMemcpyDeviceToHost);
//	printTensor(temp1,a,r2,r3);

	hipblasSgemmStridedBatched(
			
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			b,
			a,
			r2,
			&alpha,
			d_U2,
			r2,
			0,
			d_temp1,
			r2,
			r2*a,
			&beta,
			d_temp2,
			b,
			b*a,
			r3
			);

	hipMemcpy(temp2,d_temp2,sizeof(dt)*a*b*r3,hipMemcpyDeviceToHost);
	
//	printTensor(temp2,a,b,r3);

	// now temp2 store the real value 
	//we will mat3,and the 
	temp3 = tensor2mat(temp2,a,b,r3,3);
//	printTensor(temp3,r3,a*b,1);
	

	hipMemcpy(d_temp3,temp3,sizeof(dt)*r3*a*b,hipMemcpyHostToDevice);

	hipblasSgemm(
			
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			a*b,
			c,
			r3,
			&alpha,
			d_temp3,
			a*b,
			d_U3,
			r3,
			&beta,
			d_temp,
			a*b

			);

	hipMemcpy(temp,d_temp,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);

//	printTensor(temp,c,a*b,1);

	hipblasDestroy(handle);
	
	//mode3 to Tensor 
	//r3 * r1×r2   to r1*r2*r3
	
	for(int k = 0;k<c;k++){
		for(int i = 0;i<a;i++){
			for(int j = 0;j<b;j++){
				rec[k*a*b+i*b+j] = temp[k*a*b+j*a+i];
			}
		}
	}

	hipFree(d_core);
	hipFree(d_U1);
	hipFree(d_U2);
	hipFree(d_U3);
	hipFree(d_temp1);
	hipFree(d_temp2);
	hipFree(d_temp3);
	hipFree(d_temp);
//	delete[] temp1; temp1 = nullptr;
	delete[] temp2; temp2 = nullptr;
	delete[] temp3; temp3 = nullptr;
	delete[] temp; temp = nullptr;

}



void Btensor2mat(dt *A,dt *A1,dt *A2,dt *A3,int a,int b,int c){
	
	dt *d_AA;
	dt *d_A1;
	dt *d_A2;
	dt *d_A3;

	hipMalloc((void **)&d_AA,sizeof(dt)*a*b*c);
	hipMalloc((void **)&d_A1,sizeof(dt)*a*b*c);

	hipMemcpy(d_AA,A,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);

	dim3 threads(512,1,1);
	dim3 blocks(((a*b*c+512-1)/512),1,1);

	mode1tran<<<blocks,threads>>>(d_AA,d_A1,a,b,c);
	hipMemcpy(A1,d_A1,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);
	hipFree(d_A1);

	hipMalloc((void **)&d_A2,sizeof(dt)*a*b*c);
	mode2tran<<<blocks,threads>>>(d_AA,d_A2,a,b,c);
	hipMemcpy(A2,d_A2,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);
	hipFree(d_A2);

	hipMalloc((void **)&d_A3,sizeof(dt)*a*b*c);
	mode3tran<<<blocks,threads>>>(d_AA,d_A3,a,b,c);

	hipMemcpy(A3,d_A3,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost);

	hipFree(d_AA);
	hipFree(d_A3);

	cout<<"Btensor2mat is over"<<endl;

/*		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int j = 0;j<b;j++){
					t2m[i*b*c+k*b+j] = A[k*a*b+i*b+j];
				}
			}
		}
	
		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int j = 0;j<b;j++){
					t2m[j*a*c+k*a+i] = A[k*a*b+i*b+j];
				}
			}
		}

		for(int k = 0;k<c;k++){
			for(int i = 0;i<a;i++){
				for(int  j = 0;j<b;j++){
					 t2m[k*a*b+j*a+i]= A[k*a*b+i*b+j];
				}
			}
		}

*/

}

void getvector(dt *A,dt *U,int m,int n,int r){
	//we compute A*A'
	dt *d_A;
	dt *d_AT;
	hipMalloc((void**)&d_A,sizeof(dt)*m*n);
	hipMalloc((void**)&d_AT,sizeof(dt)*m*n);
	dt *d_AAT;
	hipMalloc((void**)&d_AAT,sizeof(dt)*m*m);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipMemcpy(d_A,A,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	dim3 threads(512,1,1);
	dim3 blocks((m*n+512-1)/512,1,1);
	transpose<<<blocks,threads>>>(d_A,d_AT,m,n);  // now d_AT n*m
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);

	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			m,
			m,
			n,
			&alpha,
			d_AT,
			HIP_R_32F,
			m,
			d_A,
			HIP_R_32F,
			n,
			&beta,
			d_AAT,  //store A*A'
			HIP_R_32F,
			m,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_AT);
// eig
	hipsolverHandle_t cusolverH = NULL;
	dt *V = new dt[m*m]();
	dt *V1 = new dt[r*m]();
	dt *d_W;
	int *devInfo = NULL;
	dt *d_work = NULL;
	int lwork;
	int info_gpu = 0;
	hipsolverDnCreate(&cusolverH);
	hipMalloc((void**)&devInfo,sizeof(int));
	hipMalloc((void**)&d_W,sizeof(dt)*m);
	
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverDnSsyevd_bufferSize(
			cusolverH,
			jobz,
			uplo,
			m,
			d_AAT,
			m,
			d_W,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);

	hipsolverDnSsyevd(
			cusolverH,
			jobz,
			uplo,
			m,
			d_AAT,   //store vectors
			m,
			d_W,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
	hipMemcpy(V,d_AAT,sizeof(dt)*m*m,hipMemcpyDeviceToHost);
//	hipMemcpy(W,d_W,sizeof(dt)*m,hipMemcpyDeviceToHost);
	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
	if(info_gpu == 0){
		cout<<"ok"<<endl;
	}

	hipFree(d_W);
	hipFree(d_work);
	hipFree(devInfo);
	hipFree(d_AAT);
//	printTensor(V,m,m,1);
//	printTensor(W,m,1,1);
	hipsolverDnDestroy(cusolverH);
	hipDeviceReset();
//	printTensor(V,m,m,1);
	for(int i=0;i<r;i++){
		for(int j = 0;j<m;j++){
			V1[i*m+j] = V[i*m+j+m*(m-r)];
			U[j*r+i] = V1[i*m+j];
		}
	}
//	printTensor(U,m,r,1);
		
	delete[] V;V=nullptr;
	delete[] V1;V1=nullptr;

}
void KRao(dt *X,dt *M,dt *N,dt *left,dt *right,int m,int n,int r,int k,int flag){
// m*r  n*r  m*n*r
	dt *d_M;
	hipMalloc((void **)&d_M,sizeof(dt)*m*r);
	hipMemcpy(d_M,M,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	
	dt *d_MT;
	hipMalloc((void **)&d_MT,sizeof(dt)*m*r);
	dim3 threads(512,1,1);
	dim3 blocks1((m*r+512-1)/512,1,1);
	transpose<<<blocks1,threads>>>(d_M,d_MT,m,r);

	dt *d_MTM;
	hipMalloc((void **)&d_MTM,sizeof(dt)*r*r);

	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r,
			r,
			m,
			&alpha,
			d_M,
			HIP_R_32F,
			r,
			d_MT,
			HIP_R_32F,
			m,
			&beta,
			d_MTM,
			HIP_R_32F,
			r,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	hipFree(d_MT);

	dt *d_N;
	dt *d_NT;
	hipMalloc((void **)&d_N,sizeof(dt)*n*r);
	hipMemcpy(d_N,N,sizeof(dt)*n*r,hipMemcpyHostToDevice);

	hipMalloc((void **)&d_NT,sizeof(dt)*n*r);
	dim3 blocks2((n*r+512-1)/512,1,1);
	transpose<<<blocks2,threads>>>(d_N,d_NT,n,r);
	

	//now d_MT*M  d_NT*N

	dt *d_NTN;
	hipMalloc((void**)&d_NTN,sizeof(dt)*r*r);
	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r,
			r,
			n,
			&alpha,
			d_N,
			HIP_R_32F,
			r,
			d_NT,
			HIP_R_32F,
			n,
			&beta,
			d_NTN,
			HIP_R_32F,
			r,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	hipFree(d_NT);

	dim3 blocks3((r*r+512-1)/512,1,1);
	elepro<<<blocks3,threads>>>(d_MTM,d_NTN,r);
	hipMemcpy(right,d_MTM,sizeof(dt)*r*r,hipMemcpyDeviceToHost);
	hipFree(d_MTM);
	hipFree(d_NTN);

	//right is solve the right

	dt *d_dot;
	hipMalloc((void **)&d_dot,sizeof(dt)*m*n*r);
	dim3 blocks((m*n*r+512-1)/512,1,1);
	krpro<<<blocks,threads>>>(d_M,d_N,d_dot,m,n,r);
	hipFree(d_M);
	hipFree(d_N);
	//res store the dotpro  bc*a

	dt *d_X;
	dt *d_X_M;
	hipMalloc((void**)&d_X,sizeof(dt)*m*n*k);
	hipMalloc((void**)&d_X_M,sizeof(dt)*m*n*k);
	hipMemcpy(d_X,X,sizeof(dt)*m*n*k,hipMemcpyHostToDevice);

	dim3 blocks4((m*n*k+512-1)/512,1,1);
	if(flag == 1){
		mode1tran<<<blocks4,threads>>>(d_X,d_X_M,k,n,m);
	}else if(flag == 2){
		mode2tran<<<blocks4,threads>>>(d_X,d_X_M,n,k,m);
	}else{
		mode3tran<<<blocks4,threads>>>(d_X,d_X_M,n,m,k);
	}
	hipFree(d_X);

	// d_X1*d_dot = left
	dt *d_left;
	hipMalloc((void**)&d_left,sizeof(dt)*k*r);
	hipblasGemmEx(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			r,
			k,
			m*n,
			&alpha,
			d_dot,
			HIP_R_32F,
			r,
			d_X_M,
			HIP_R_32F,
			m*n,
			&beta,
			d_left,
			HIP_R_32F,
			r,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
			);
	hipMemcpy(left,d_left,sizeof(dt)*k*r,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_left);
	hipFree(d_X_M);
	hipFree(d_dot);

}



void solve(dt *left,dt *right,dt *res,int r,int m){
	dt *d_left;
	dt *d_right;
	hipMalloc((void**)&d_right,sizeof(dt)*m*r);
	hipMalloc((void**)&d_left,sizeof(dt)*r*r);
	dt *d_work;
	int *d_info;
	int lwork;
	hipsolverHandle_t handle;
	hipsolverDnCreate(&handle);
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipMalloc((void**)&d_info,sizeof(int));

	hipMemcpy(d_left,left,sizeof(dt)*r*r,hipMemcpyHostToDevice);
	hipMemcpy(d_right,right,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	hipsolverDnSpotrf_bufferSize(
			handle,
			uplo,
			r,
			d_left,
			r,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);

	hipsolverDnSpotrf(
			handle,
			uplo,
			r,
			d_left,
			r,
			d_work,
			lwork,
			d_info
			);
	hipsolverDnSpotrs(
			handle,
			uplo,
			r,
			m,
			d_left,
			r,
			d_right,
			r,
			d_info
			);
	hipDeviceSynchronize();
//	int info_gpu;
//	hipMemcpy(&info_gpu,d_info,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(res,d_right,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
/*	if(info_gpu == 0){
		cout<<"OK"<<endl;
		cout<<endl;
	}
*/
// d_right store the A/B/C m*r


//	printTensor(res,m,r,1);
	dt *sum = new dt[r]();
	for(int i = 0;i<r;i++){
		for(int j = 0;j<m;j++){
				sum[i] += res[j*r+i]*res[j*r+i]; 
			}
		}

//	printTensor(sum,r,1,1);
	dt *d_sum;
	hipMalloc((void**)&d_sum,sizeof(dt)*r);
	hipMemcpy(d_sum,sum,sizeof(dt)*r,hipMemcpyHostToDevice);
	dim3 threads(512,1,1);
	dim3 blocks((m*r+512-1)/512,1,1);
	norm<<<blocks,threads>>>(d_right,d_sum,m,r);
	hipMemcpy(res,d_right,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
	hipFree(d_sum);
	hipFree(d_left);
	hipFree(d_right);
	hipFree(d_info);
	hipFree(d_work);
	hipsolverDnDestroy(handle);
	hipDeviceReset();
	delete[] sum;sum = nullptr;


}

/*dt norm(dt *X,int a,int b,int c){
	dt temp = 0.0;
	dt *d_X;
	dt d_temp;
	hipMalloc(d_X,sizeof(dt)*a*b*c);
	hipMalloc(d_temp,sizeof(dt));
	hipMemcpy(d_X,X,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	dim3 threads(512,1,1);
	dim3 blocks((a*b*c+512-1)/512,1,1);
	Norm<<<blocks,threads>>>(d_X,d_temp,a,b,c);
	hipMemcpy(temp,d_temp)

	
}*/

/*void recontr(dt *src,dt *des,dt *A,dt *B,dt *C,int a,int b,int c,int r){
	dt *d_C;
	dt *d_B;
	dt *d_A;
	dt *d_des;
	dt *d_src;
	dt *d_BA;

	hipMalloc(d_B,sizeof(dt)*b*r);
	hipMalloc(d_A,sizeof(dt)*a*r);
	hipMalloc(d_CB,sizeof(dt)*a*b*r);
	hipMemcpy(d_B,B,sizeof(dt)*b*r);
	hipMemcpy(d_A,A,sizeof(dt)*a*r);


}*/

void cp_als(dt *X,dt *A,dt *B,dt *C,int a,int b,int c,int r){

/*	dt *X_temp = new dt[a*b*c]();
	dt *error = new dt[100];
	dt tol = 1e06;
	dt X_norm = 0.0;
	for(int i = 0;i<a*b*c;i++){
		X_norm += X[i]*X[i];
	}
	X_norm = sqrt(X_norm);
*/
	dt *temp1 = new dt[a*r]();
	dt *temp2 = new dt[b*r]();
	dt *temp3 = new dt[c*r]();
	dt *tem1 = new dt[r*r]();
	dt *tem2 = new dt[r*r]();
	dt *tem3 = new dt[r*r]();
	
	for(int i = 0;i<1;i++){

		KRao(X,C,B,temp1,tem1,c,b,r,a,1);
		solve(tem1,temp1,A,r,a);     // we get A  

		KRao(X,C,A,temp2,tem2,c,a,r,b,2);
		solve(tem2,temp2,B,r,b);     // we get B
		
		KRao(X,B,A,temp3,tem3,b,a,r,c,3);
		solve(tem3,temp3,C,r,c);    //we get C

//		recontr(X,X_temp,A,B,C,a,b,c,r);

//		error[i] = 
	}

	delete[] temp1;temp1 = nullptr;
	delete[] temp2;temp1 = nullptr;
	delete[] temp3;temp1 = nullptr;
	delete[] tem1;tem1 = nullptr;
	delete[] tem2;tem2 = nullptr;
	delete[] tem3;tem3 = nullptr;
}
